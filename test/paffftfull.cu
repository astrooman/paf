#include "hip/hip_runtime.h"
/***********************************
VERSION OF THE FFT ALGORITHM FOR PAF
WITH FFT, POWER, AND AVERAGING ADDED

GENERAL TIMING RESULTS:
(E - EVENTS, P - NVPROF)


BOTH RESULTS QUOTED AS THERE ARE
SOME DISAGREEMENTS ON WHICH ONE IS
THE MOST RELIABLE ESTIMATE
***********************************/

#include <chrono>
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <stdlib.h>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <nvToolsExt.h>

using std::cerr;
using std::cout;
using std::endl;
using std::string;


static const uint32_t colors[] = { 0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff, 0x0000ffff, 0x00ff0000, 0x00ffffff, 0x00fd482f };
static const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
	int color_id = cid; \
	color_id = color_id%num_colors;\
	nvtxEventAttributes_t eventAttrib = {0}; \
	eventAttrib.version = NVTX_VERSION; \
	eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
	eventAttrib.colorType = NVTX_COLOR_ARGB; \
	eventAttrib.color = colors[color_id]; \
	eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
	eventAttrib.message.ascii = name; \
	nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();

#define MEMALIGN 4096

template <typename T>
void geterror(T res, std::string place)
{
    if ( (res != HIPFFT_SUCCESS) && (res != hipSuccess) )
        cout << "Error in " << place << "!! Error: " << res << endl;
}

void printhelp(void);

// GPU kernel
// need to figure out how to store the data efficienty
// don't want to introduce offset memory access
// which will significantly decrease the effective bandwidth

// of version with offset memory access
__global__ void poweraddkof(hipfftComplex *arr_in, float *arr_out, unsigned int size)
{

	int index1 = blockIdx.x * blockDim.x + threadIdx.x;
	// offset introduced - can cause some slowing down
	int index2 = blockIdx.x * blockDim.x + threadIdx.x + size;

	if (index1 < size) {
		float power1 = arr_in[index1].x * arr_in[index1].x + arr_in[index1].y * arr_in[index1].y;
		float power2 = arr_in[index2].x * arr_in[index2].x + arr_in[index2].y * arr_in[index2].y;
		arr_out[index1] = (power1 + power2) / 2;
	}
}

int main(int argc, char* argv[])
{
	bool preinit = true;
	bool usekernel = true;
	string mode = "n";
    if (argc >= 2) {
		for (int ii = 0; ii < argc; ii++) {
			if (string(argv[ii]) == "-h") {
				printhelp();
			} else if (string(argv[ii]) == "-m") {
				ii++;
				mode = string(argv[ii]);
			} else if (string(argv[ii]) == "-p") {
				preinit = false;
			} else if (string(argv[ii]) == "-t") {
				usekernel = false;
			}
    		}
    }

	// that must be done before any CUDA context is created
	if (mode == "m")
		hipSetDeviceFlags(hipDeviceMapHost);

	if (preinit) {

		cout << "Pre-initialisation...\n";
    	PUSH_RANGE("FFT pre-init", 0)
    	// this should make the first proper FFT execution faster
    	hipfftHandle preinit;
    	geterror(hipfftPlan1d(&preinit, 32, HIPFFT_C2C, 1), "init plan make");
    	POP_RANGE

	}

	// this stuff will stay the same between runs
	// const so I don't change it by mistake at some point
	const unsigned int arrsize = 32;
    const unsigned int fftsize = arrsize;
    const unsigned int batchsize = 1152;    // the number of FFTs we want to perform at once
	const unsigned int timesamp = 2;	// will need to process more than one timesamples for averaging
	const unsigned int fullsize = fftsize * batchsize * timesamp;
	const unsigned int memsize = fullsize * sizeof(hipfftComplex);
	// limit is 1024 threads per block on all compute capablities
	// warp size is 32 on all compute capabilities
	unsigned int nthreads = 128;
	unsigned int nblocks = (fullsize / timesamp - 1) / nthreads + 1;
    // complex voltage goes in
	hipfftComplex *h_inarray = new hipfftComplex[fullsize];
	// time-averaged power goes out
	float *h_outarray = new float[fullsize / timesamp];
    int sizes[1] = {fftsize};
	unsigned long seed = std::chrono::system_clock::now().time_since_epoch().count();
	std::mt19937_64 arreng{seed};
	std::normal_distribution<float> arrdis(0.0, 1.0);

	for (int ii = 0; ii < fullsize; ii++) {
			h_inarray[ii].x = arrdis(arreng);
			h_inarray[ii].y = arrdis(arreng);
	}

	if (mode == "n") {

		cout << "Will use standard memory copies...\n";

		hipfftComplex *d_inarray;
		geterror(hipMalloc((void**)&d_inarray, memsize), "device out malloc");
		// make sure memsize is even, i.e. timesamp is even
		// need only half of the original size for data averaged in time
		float *d_outarray;
		geterror(hipMalloc((void**)&d_outarray, fullsize / timesamp * sizeof(float)), "device out malloc");

		PUSH_RANGE("Multi FFT init", 1)
		hipfftHandle multiplan;
		geterror(hipfftPlanMany(&multiplan, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize * timesamp), "default plan make");
		POP_RANGE

		// time everything, together with memory copies
		PUSH_RANGE("Multi FFT exec", 2)
		geterror(hipMemcpy(d_inarray, h_inarray, memsize, hipMemcpyHostToDevice), "HtD copy");
		geterror(hipfftExecC2C(multiplan, d_inarray, d_inarray, HIPFFT_FORWARD), "default execution");
		poweraddkof<<<nblocks, nthreads>>>(d_inarray, d_outarray, fullsize / 2);
		geterror(hipGetLastError(), "default kernel exec");
		geterror(hipMemcpy(h_outarray, d_outarray, fullsize / timesamp * sizeof(float), hipMemcpyDeviceToHost), "DtH copy");
		POP_RANGE

		geterror(hipfftDestroy(multiplan), "default plan destroy");
		geterror(hipFree(d_inarray), "device in free");
		geterror(hipFree(d_outarray), "device out free");

	} else if (mode == "p") {

		cout << "Will use pinned memory";

		hipStream_t stream1, stream2;
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);

		hipfftComplex *h_inp, *d_inp;
		hipfftComplex *h_inp2, *d_inp2;
		float *h_outp, *d_outp;
		float *h_outp2, *d_outp2;

		// make sure the size is a multiple of the page size
		// no need to actually use all of this memory in calculations
		// might not be necessary with out data as we end up with multiples of 4096
		// but better leave to be one the safe side
		int alignsizein = ((int)((int)memsize + MEMALIGN - 1) / MEMALIGN) * MEMALIGN;
		int alignsizeout = ((int)((int)(fullsize / 2) * sizeof(float) + MEMALIGN -1) / MEMALIGN) * MEMALIGN;

		cout << "Original in size: " << memsize << "B\n";
		cout << "Page multiple in size: " << alignsizein << "B\n";
		cout << "Page multiple out size: " << alignsizeout << "B\n";

		hipfftHandle plan1, plan2;
		hipfftPlanMany(&plan1, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize * timesamp);
		hipfftPlanMany(&plan2, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize * timesamp);

		hipfftSetStream(plan1, stream1);
		hipfftSetStream(plan2, stream2);

		posix_memalign((void**)&h_inp, MEMALIGN, alignsizein);
		posix_memalign((void**)&h_inp2, MEMALIGN, alignsizein);
		posix_memalign((void**)&h_outp, MEMALIGN, alignsizeout);
		posix_memalign((void**)&h_outp2, MEMALIGN, alignsizeout);

		for (int ii = 0; ii < fullsize; ii++) {
			h_inp[ii].x = ii;
			h_inp[ii].y = 2 * ii;

			h_inp2[ii].x = ii;
			h_inp2[ii].y = 2 * ii;


                        //h_inp[ii].x = arrdis(arreng);
                        //h_inp[ii].y = arrdis(arreng);
        	}

		hipHostRegister(h_inp, alignsizein, hipHostRegisterDefault);
		hipHostRegister(h_inp2, alignsizein, hipHostRegisterDefault);
		hipHostRegister(h_outp, alignsizeout, hipHostRegisterDefault);
		hipHostRegister(h_outp2, alignsizeout, hipHostRegisterDefault);

		hipHostGetDevicePointer((void**)&d_inp, (void*)h_inp, 0);
		hipHostGetDevicePointer((void**)&d_outp, (void*)h_outp, 0);
		hipHostGetDevicePointer((void**)&d_inp2, (void*)h_inp2, 0);
		hipHostGetDevicePointer((void**)&d_outp2, (void*)h_outp2, 0);

		hipfftExecC2C(plan1, d_inp, d_inp, HIPFFT_FORWARD);
		hipfftExecC2C(plan2, d_inp2, d_inp2, HIPFFT_FORWARD);

		poweraddkof<<<nblocks, nthreads, 0, stream1>>>(d_inp, d_outp, fullsize / 2);
		poweraddkof<<<nblocks, nthreads, 0, stream2>>>(d_inp2, d_outp2, fullsize / 2);

		if(hipGetLastError() != hipSuccess)
			cout << "Error!!" << endl;

		hipHostUnregister(h_inp);
		hipHostUnregister(h_outp);
		hipHostUnregister(h_inp2);
		hipHostUnregister(h_outp2);

		free(h_inp);
		free(h_outp);
		free(h_inp2);
		free(h_outp2);

	} else if (mode == "m") {

		cout << "Will use mapped pinned memory...\n";

		hipfftComplex *h_inarraym, *d_inarray;
		float *h_outarraym, *d_outarray;
		geterror(hipHostAlloc((void**)&h_inarraym, memsize, hipHostMallocMapped), "host in alloc");
		geterror(hipHostAlloc((void**)&h_outarraym, fullsize / timesamp * sizeof(float), hipHostMallocMapped), "host out alloc");
		geterror(hipHostGetDevicePointer((void**)&d_inarray, (void*)h_inarraym, 0), "in dev pointer");
		geterror(hipHostGetDevicePointer((void**)&d_outarray, (void*)h_outarraym, 0), "out dev pointer");

		for (int ii = 0; ii < fullsize; ii++) {
				h_inarraym[ii].x = arrdis(arreng);
				h_inarraym[ii].y = arrdis(arreng);
		}

		PUSH_RANGE("Multi mapped FFT init", 1)
		hipfftHandle multiplan;
		geterror(hipfftPlanMany(&multiplan, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize * timesamp), "mapped plan make");
		POP_RANGE

		PUSH_RANGE("Multi mapped FFT exec", 2)
		geterror(hipfftExecC2C(multiplan, d_inarray, d_inarray, HIPFFT_FORWARD), "mapped execution");
		poweraddkof<<<nblocks, nthreads>>>(d_inarray, d_outarray, fullsize / 2);
		geterror(hipGetLastError(), "mapped kernel exec");
		POP_RANGE

		geterror(hipfftDestroy(multiplan), "mapped plan destroy");
		geterror(hipHostFree(h_inarraym), "host in free");
		geterror(hipHostFree(h_outarraym), "host out free");

		

	} else if (mode == "a") {

		cout << "Will use asynchronous memory copies...\n";

		// 2 streams should be enough, or might not show any benefit at all
		hipStream_t stream1, stream2;
		hipStreamCreate(&stream1);
		hipStreamCreate(&stream2);

		hipfftHandle multiplans1, multiplans2;
		geterror(hipfftPlanMany(&multiplans1, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize * timesamp), "async plan make 1");
		geterror(hipfftPlanMany(&multiplans2, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize * timesamp), "async plan make 2");

		geterror(hipfftSetStream(multiplans1, stream1), "FFT set stream 1");
		geterror(hipfftSetStream(multiplans2, stream2), "FFT set stream 2");

		// must use pinned memory
		// very crude memory managment here
		// this will end up in loops at the end of the day
		// I will assume this stuff executes OK
		hipfftComplex *h_inarraya1, *d_inarray1, *h_inarraya2, *d_inarray2;
		float *h_outarraya1, *d_outarray1, *h_outarraya2, *d_outarray2;
		hipHostAlloc((void**)&h_inarraya1, memsize, hipHostMallocDefault);
		hipHostAlloc((void**)&h_inarraya2, memsize, hipHostMallocDefault);

		for (int ii = 0; ii < fullsize; ii++) {
				h_inarraya1[ii].x = arrdis(arreng);
				h_inarraya1[ii].y = arrdis(arreng);
				h_inarraya2[ii].x = arrdis(arreng);
				h_inarraya2[ii].y = arrdis(arreng);
		}

		hipHostAlloc((void**)&h_outarraya1, fullsize / timesamp * sizeof(float), hipHostMallocDefault);
		hipHostAlloc((void**)&h_outarraya2, fullsize / timesamp * sizeof(float), hipHostMallocDefault);
		hipMalloc((void**)&d_inarray1, memsize);
		hipMalloc((void**)&d_inarray2, memsize);
		hipMalloc((void**)&d_outarray1, fullsize / timesamp * sizeof(float));
		hipMalloc((void**)&d_outarray2, fullsize / timesamp * sizeof(float));

		geterror(hipMemcpyAsync(d_inarray1, h_inarraya1, memsize, hipMemcpyHostToDevice, stream1), "HtD async copy 1");
		geterror(hipMemcpyAsync(d_inarray2, h_inarraya2, memsize, hipMemcpyHostToDevice, stream2), "HtD async copy 2");

		geterror(hipfftExecC2C(multiplans1, d_inarray1, d_inarray1, HIPFFT_FORWARD), "async execution 1");
		geterror(hipfftExecC2C(multiplans2, d_inarray2, d_inarray2, HIPFFT_FORWARD), "async execution 2");

		poweraddkof<<<nblocks, nthreads, 0, stream1>>>(d_inarray1, d_outarray1, fullsize / 2);
		poweraddkof<<<nblocks, nthreads, 0, stream2>>>(d_inarray2, d_outarray2, fullsize / 2);

		geterror(hipMemcpyAsync(h_outarraya1, d_outarray1, fullsize / timesamp * sizeof(float), hipMemcpyDeviceToHost, stream1), "DtH async copy 1");
		geterror(hipMemcpyAsync(h_outarraya2, d_outarray2, fullsize / timesamp * sizeof(float), hipMemcpyDeviceToHost, stream2), "DtH async copy 2");

		// I will assume this stuff executes OK as well
		hipStreamDestroy(stream1);
		hipStreamDestroy(stream2);

		hipFree(d_inarray1);
		hipFree(d_inarray2);
		hipFree(d_outarray1);
		hipFree(d_outarray2);
		hipHostFree(h_inarraya1);
		hipHostFree(h_inarraya2);
		hipHostFree(h_outarraya1);
		hipHostFree(h_outarraya2);

	} else {
		cout << "Invalid memory mode option!! Will now quit!!";
	}

	geterror(hipfftDestroy(preinit), "init plan destroy");
	delete [] h_inarray;
	delete [] h_outarray;

    hipDeviceReset();

    return 0;

}

void printhelp(void)
{
	cout << "Test code for PAF FFT code" << endl << endl;
	cout << "Available options:" << endl;
	cout << "\t-p - switch pre-initialisation off" << endl;
	cout << "\t-t - use Thrust functions instead of custom kernels for power and averaging" << endl;
	cout << "\t-m - memory mode: n (default) - use hipMemcpy()" << endl;
	cout << "\t\tp - use pinned memory, m - use mapped pinned memory, a - use asynchronous copies" << endl;

}
