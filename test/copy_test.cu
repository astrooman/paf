#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>

#include <hipfft/hipfft.h>

using std::cout;
using std::endl;

#define XSIZE = 7
#define YSIZE = 128
#define ZSIZE = 48

#define cudaCheckError(myerror) {checkGPU((myerror), __FILE__, __LINE__);}

inline void checkGPU(hipError_t code, const char *file, int line) {

    if (code != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(code) << " in file " << file << ", line " << line << std::endl;
        exit(EXIT_FAILURE);
        // TODO: throw exception instead of exiting
    }

}

int main(int argc, char *argv[])
{

    float alloc_elapsed;
    hipEvent_t alloc_start;
    hipEvent_t alloc_end;

    hipEventCreate(&alloc_start);
    hipEventCreate(&alloc_end);

    char *h_in = new char[8 * XSIZE * YSIZE * ZSIZE];
    for (int ii = 0; ii < 8 * XSIZE * YSIZE * ZSIZE)
        h_in[ii] = ii % 7;

    hipChannelFormatDesc cdesc;
    hipExtent volume;
    hipMemcpy3DParms params = {0};

    cdesc = hipCreateChannelDesc<int2>();
    cudaCheckError(hipPeekAtLastError());
    volume = make_hipExtent(XSIZE, YSIZE, ZSIZE);
    cudaCheckError(hipPeekAtLastError());
    hipArray *d_array;

    hipEventRecord(alloc_start, 0);
    cudaCheckError(hipMalloc3DArray(&d_array, &cdesc, volume));
    cudaCheckError(hipBindTextureToArray(tex, d_array));
    hipEventRecord(alloc_end, 0);
    hipEventSynchronize(alloc_end);
    hipEventElapsedTime(&alloc_elapsed, alloc_start, alloc_end);

    cout << "3D alloc: " << alloc_elapsed << "ms" << endl;

    params.extent = volume;
    params.dstArray = d_array;
    params.kind = hipMemcpyHostToDevice;
    params.srcPtr = make_hipPitchedPtr((void*)h_in, XSIZE * 8, XSIZE * 8, YSIZE);

    tex.filterMode = hipFilterModePoint;
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.addressMode[2] = hipAddressModeClamp;

    float copy_elapsed;
    hipEvent_t copy_start;
    hipEvent_t copy_end;

    hipEventCreate(&copy_start);
    hipEventCreate(&copy_end);

    hipEventRecord(copy_start,0);
    cudaCheckError(hipMemcpy3D(&params));
    cudaEventrecord(copy_end.0);
    hipEventSynchronize(copy_end);
    hipEventElapsedTime(&copy_elapsed, copy_start, copy_end);

    cout << "3D memcpy: " << copy_elapsed << "ms" << endl;

    hipDeviceSynchronize();
    hipUnbindTexture(tex);

    hipFreeArray(d_array);
    hipFree(d_out);
    delete [] h_in;

    return 0;
}
