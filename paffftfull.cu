/***********************************
VERSION OF THE FFT ALGORITHM FOR PAF
WITH FFT, POWER, AND AVERAGING ADDED

GENERAL TIMING RESULRS:
(E - EVENTS, P - NVPROF)


BOTH RESULTS QUOTED AS THERE ARE
SOME DISAGREEMENTS ON WHICH ONE IS
THE MOST RELIABLE ESTIMATE
***********************************/

#include <chrono>
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <nvToolsExt.h>

using std::cerr;
using std::cout;
using std::endl;


static const uint32_t colors[] = { 0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff, 0x0000ffff, 0x00ff0000, 0x00ffffff, 0x00fd482f };
static const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
	int color_id = cid; \
	color_id = color_id%num_colors;\
	nvtxEventAttributes_t eventAttrib = {0}; \
	eventAttrib.version = NVTX_VERSION; \
	eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
	eventAttrib.colorType = NVTX_COLOR_ARGB; \
	eventAttrib.color = colors[color_id]; \
	eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
	eventAttrib.message.ascii = name; \
	nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();

void geterror(hipfftResult res, std::string place);

int main(int argc, char* argv[])
{

    if (argc >= 2) {


    }


    const unsigned int arrsize = 32;
    const unsigned int fftsize = arrsize;
    const unsigned int batchsize = 1152;    // the number of FFTs we want to perform at once
    hipfftComplex *h_inarray = new hipfftComplex[arrsize];
    int sizes[1] = {fftsize};


    cout << "Pre-initialisation...\n";

    PUSH_RANGE("FFT pre-init", 0)
    // this should make the first proper FFT execution faster
    hipfftHandle preinit;
    hipfftPlan1d(&preinit, fftsize, HIPFFT_C2C, 1);
    POP_RANGE

    unsigned long seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::mt19937_64 arreng{seed};
    std::normal_distribution<float> arrdis(0.0, 1.0);

    for (int ii = 0; ii < arrsize; ii++) {
            h_inarray[ii].x = arrdis(arreng);
            h_inarray[ii].y = arrdis(arreng);
    }

    hipfftComplex *d_inarray;
    hipMalloc((void**)&d_inarray, sizeof(hipfftComplex) * arrsize);
    hipMemcpy(d_inarray, h_inarray, sizeof(hipfftComplex) * arrsize, hipMemcpyHostToDevice);

    cout << "Performing single FFT...\n";

    hipProfilerStart();

    PUSH_RANGE("Single FFT init", 1)
    hipfftHandle singleplan;
    geterror(hipfftPlan1d(&singleplan, fftsize, HIPFFT_C2C, 1), "single FFT plan");
    POP_RANGE

    PUSH_RANGE("Single FFT exec", 2)
    geterror(hipfftExecC2C(singleplan, d_inarray, d_inarray, HIPFFT_FORWARD), "single FFT execution");
    POP_RANGE

    hipfftDestroy(singleplan);
    hipFree(d_inarray);
    delete [] h_inarray;

    hipfftComplex *h_inarraym = new hipfftComplex[arrsize * batchsize];

    for (int ii = 0; ii < arrsize * batchsize; ii++) {
        h_inarraym[ii].x = arrdis(arreng);
        h_inarraym[ii].y = arrdis(arreng);

    }

    hipfftComplex *d_inarraym;
    hipMalloc((void**)&d_inarraym, sizeof(hipfftComplex) * arrsize * batchsize);
    hipMemcpy(d_inarraym, h_inarraym, sizeof(hipfftComplex) * arrsize * batchsize, hipMemcpyHostToDevice);

    PUSH_RANGE("Multi FFT init", 3)
    hipfftHandle multiplan;
    geterror(hipfftPlanMany(&multiplan, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize), "multi FFT plan");
    POP_RANGE

    PUSH_RANGE("Multi FFT exec", 4)
    geterror(hipfftExecC2C(multiplan, d_inarraym, d_inarraym, HIPFFT_FORWARD), "multi FFT execution");
    POP_RANGE

    hipfftDestroy(multiplan);
    hipFree(d_inarraym);
    delete [] h_inarraym;

    unsigned int timesamp = 1;

    hipfftComplex *h_inarraym2 = new hipfftComplex[arrsize * batchsize * timesamp];

    for (int ii = 0; ii < arrsize * batchsize * timesamp; ii++) {
        h_inarraym2[ii].x = arrdis(arreng);
        h_inarraym2[ii].y = arrdis(arreng);

    }

    hipfftComplex *d_inarraym2;
    hipMalloc((void**)&d_inarraym2, sizeof(hipfftComplex) * arrsize * batchsize * timesamp);
    hipMemcpy(d_inarraym2, h_inarraym2, sizeof(hipfftComplex) * arrsize * batchsize * timesamp, hipMemcpyHostToDevice);

    PUSH_RANGE("Multi FFT 2 init", 5)
    hipfftHandle multi2plan;
    geterror(hipfftPlanMany(&multi2plan, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize * timesamp), "multi FFT 2 plan");
    POP_RANGE

    PUSH_RANGE("Multi FFT 2 exec", 6)
    geterror(hipfftExecC2C(multi2plan, d_inarraym2, d_inarraym2, HIPFFT_FORWARD), "multi FFT 2 execution");
    POP_RANGE

    hipfftDestroy(multi2plan);
    hipFree(d_inarraym2);
    delete [] h_inarraym2;

    cout << "Use different timing methods...\n";

    hipfftComplex *h_inarraym3 = new hipfftComplex[arrsize * batchsize * timesamp];

    for (int ii = 0; ii < arrsize * batchsize * timesamp; ii++) {
        h_inarraym3[ii].x = arrdis(arreng);
        h_inarraym3[ii].y = arrdis(arreng);

    }

    hipfftComplex *d_inarraym3;
    hipMalloc((void**)&d_inarraym3, sizeof(hipfftComplex) * arrsize * batchsize * timesamp);
    hipMemcpy(d_inarraym3, h_inarraym3, sizeof(hipfftComplex) * arrsize * batchsize * timesamp, hipMemcpyHostToDevice);

    hipEvent_t init_start, init_end, exec_start, exec_end;
    hipEventCreate(&init_start);
    hipEventCreate(&init_end);
    hipEventCreate(&exec_start);
    hipEventCreate(&exec_end);

    hipEventRecord(init_start);
    hipfftHandle multi3plan;
    geterror(hipfftPlanMany(&multi3plan, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize * timesamp), "multi FFT 3 plan");
    hipEventRecord(init_end);

    hipEventRecord(exec_start);
    geterror(hipfftExecC2C(multi3plan, d_inarraym3, d_inarraym3, HIPFFT_FORWARD), "multi FFT 3 execution");
    hipEventRecord(exec_end);

    float init_time, exec_time;

    hipEventElapsedTime(&init_time, init_start, init_end);
    hipEventElapsedTime(&exec_time, exec_start, exec_end);

    cout << "Init time: " << init_time << "ms\n";
    cout << "Exec time: " << exec_time << "ms\n";

    hipfftDestroy(multi3plan);
    hipFree(d_inarraym3);
    delete [] h_inarraym3;

    // try slightly different approach to memory to limit HtoD time
    hipSetDeviceFlags(hipDeviceMapHost);

    hipfftComplex *h_inarraym4;
    hipfftComplex *d_inarraym4;

    hipHostAlloc((void**)&h_inarraym4, sizeof(hipfftComplex) * arrsize * batchsize, hipHostMallocMapped);
    hipHostGetDevicePointer((void**)&d_inarraym4, (void *)h_inarraym4, 0);

    for (int ii = 0; ii < arrsize * batchsize; ii++) {
        h_inarraym4[ii].x = arrdis(arreng);
        h_inarraym4[ii].y = arrdis(arreng);

    }

    hipEventRecord(init_start);
    hipfftHandle multi4plan;
    geterror(hipfftPlanMany(&multi4plan, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize), "multi FFT 4 plan");
    hipEventRecord(init_end);

    hipEventRecord(exec_start);
    geterror(hipfftExecC2C(multi4plan, d_inarraym4, d_inarraym4, HIPFFT_FORWARD), "multi FFT 4 execution");
    hipEventRecord(exec_end);

    hipEventElapsedTime(&init_time, init_start, init_end);
    hipEventElapsedTime(&exec_time, exec_start, exec_end);

    cout << "Init time: " << init_time << "ms\n";
    cout << "Exec time: " << exec_time << "ms\n";

    hipHostFree(h_inarraym4);
    hipfftDestroy(multi4plan);
    hipEventDestroy(init_start);
    hipEventDestroy(init_end);
    hipEventDestroy(exec_start);
    hipEventDestroy(exec_end);

    hipDeviceReset();

    return 0;

}

void geterror(hipfftResult res, std::string place)
{
    if (res != HIPFFT_SUCCESS)
        cout << "Error in " << place << "!! Error: " << res << endl;
}
