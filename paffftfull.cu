#include "hip/hip_runtime.h"
/***********************************
VERSION OF THE FFT ALGORITHM FOR PAF
WITH FFT, POWER, AND AVERAGING ADDED

GENERAL TIMING RESULTS:
(E - EVENTS, P - NVPROF)


BOTH RESULTS QUOTED AS THERE ARE
SOME DISAGREEMENTS ON WHICH ONE IS
THE MOST RELIABLE ESTIMATE
***********************************/

#include <chrono>
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <nvToolsExt.h>

using std::cerr;
using std::cout;
using std::endl;
using std::string;


static const uint32_t colors[] = { 0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff, 0x0000ffff, 0x00ff0000, 0x00ffffff, 0x00fd482f };
static const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
	int color_id = cid; \
	color_id = color_id%num_colors;\
	nvtxEventAttributes_t eventAttrib = {0}; \
	eventAttrib.version = NVTX_VERSION; \
	eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
	eventAttrib.colorType = NVTX_COLOR_ARGB; \
	eventAttrib.color = colors[color_id]; \
	eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
	eventAttrib.message.ascii = name; \
	nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();

template <typename T>
void geterror(T res, std::string place)
{
    if ( (res != HIPFFT_SUCCESS) && (res != hipSuccess) )
        cout << "Error in " << place << "!! Error: " << res << endl;
}

void printhelp(void);

// GPU kernel
// need to figure out how to store the data efficienty
// don't want to introduce offset memory access
// which will significantly decrease the effective bandwidth

// st version with offset memory access
__global__ void poweraddkof(hipfftComplex *arr_in, float *arr_out, unsigned int size)
{

	int index1 = blockIdx.x * blockDim.x + threadIdx.x;
	int index2 = blockIdx.x * blockDim.x + threadIdx.x + size;

	if (index1 < size) {
		float power1 = arr_in[index1].x * arr_in[index1].x + arr_in[index1].y * arr_in[index1].y;
		float power2 = arr_in[index2].x * arr_in[index2].x + arr_in[index2].y * arr_in[index2].y;
		arr_out[index1] = (power1 + power2) / 2;
	}
}

int main(int argc, char* argv[])
{
	bool preinit = true;
	bool usekernel = true;
	string mode = "n";
    if (argc >= 2) {
		for (int ii = 0; ii < argc; ii++) {
			if (string(argv[ii]) == "-h") {
				printhelp();
			} else if (string(argv[ii]) == "-m") {
				ii++;
				mode = string(argv[ii]);
			} else if (string(argv[ii]) == "-p") {
				preinit = false;
			} else if (string(argv[ii]) == "-t") {
				usekernel = false;
			}
    		}
    }

	// that must be done before any CUDA context is created
	if (mode == "m")
		hipSetDeviceFlags(hipDeviceMapHost);

	if (preinit) {

		cout << "Pre-initialisation...\n";
    	PUSH_RANGE("FFT pre-init", 0)
    	// this should make the first proper FFT execution faster
    	hipfftHandle preinit;
    	geterror(hipfftPlan1d(&preinit, 32, HIPFFT_C2C, 1), "init plan make");
    	POP_RANGE

	}

	// this stuff will stay the same between runs
	// const so I don't change it by mistake at some point
	const unsigned int arrsize = 32;
    const unsigned int fftsize = arrsize;
    const unsigned int batchsize = 1152;    // the number of FFTs we want to perform at once
	const unsigned int timesamp = 2;	// will need to process more than one timesamples for averaging
	const unsigned int fullsize = fftsize * batchsize * timesamp;
	const unsigned int memsize = fullsize * sizeof(hipfftComplex);
	// limit is 1024 threads per block on all compute capablities
	// warp size is 32 on all compute capabilities
	unsigned int nthreads = 256;
	unsigned int nblocks = (fullsize / timesamp - 1) / nthreads + 1;
    // complex voltage goes in
	hipfftComplex *h_inarray = new hipfftComplex[fullsize];
	// time-averaged power goes out
	float *h_outarray = new float[fullsize / timesamp];
    int sizes[1] = {fftsize};
	unsigned long seed = std::chrono::system_clock::now().time_since_epoch().count();
	std::mt19937_64 arreng{seed};
	std::normal_distribution<float> arrdis(0.0, 1.0);

	for (int ii = 0; ii < fullsize; ii++) {
			h_inarray[ii].x = arrdis(arreng);
			h_inarray[ii].y = arrdis(arreng);
	}

	if (mode == "n") {

		cout << "Will use standard memory copies...\n";

		hipfftComplex *d_inarray;
		geterror(hipMalloc((void**)&d_inarray, memsize), "device out malloc");
		// make sure memsize is even, i.e. timesamp is even
		// need only half of the original size for data averaged in time
		float *d_outarray;
		geterror(hipMalloc((void**)&d_outarray, fullsize / timesamp * sizeof(float)), "device out malloc");

		PUSH_RANGE("Multi FFT init", 1)
		hipfftHandle multiplan;
		geterror(hipfftPlanMany(&multiplan, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize), "default plan make");
		POP_RANGE

		// time everything, together with memory copies
		PUSH_RANGE("Multi FFT exec", 2)
		geterror(hipMemcpy(d_inarray, h_inarray, memsize, hipMemcpyHostToDevice), "HtD copy");
		geterror(hipfftExecC2C(multiplan, d_inarray, d_inarray, HIPFFT_FORWARD), "default execution");
		poweraddkof<<<nblocks, nthreads>>>(d_inarray, d_outarray, fullsize / timesamp);
		geterror(hipGetLastError(), "default kernel exec");
		geterror(hipMemcpy(h_outarray, d_outarray, fullsize / timesamp * sizeof(float), hipMemcpyDeviceToHost), "DtH copy");
		POP_RANGE

		geterror(hipfftDestroy(multiplan), "default plan destroy");
		geterror(hipFree(d_inarray), "device in free");
		geterror(hipFree(d_outarray), "device out free");

	} else if (mode == "p") {

		cout << "Will use pinned memory";

	} else if (mode == "m") {

		cout << "Will use mapped pinned memory...\n";

		hipfftComplex *h_inarraym, *d_inarray;
		float *h_outarraym, *d_outarray;
		geterror(hipHostAlloc((void**)&h_inarraym, memsize, hipHostMallocMapped), "host in alloc");
		geterror(hipHostAlloc((void**)&h_outarraym, memsize / timesamp, hipHostMallocMapped), "host out alloc");
		geterror(hipHostGetDevicePointer((void**)&d_inarray, (void*)h_inarraym, 0), "in dev pointer");
		geterror(hipHostGetDevicePointer((void**)&d_outarray, (void*)h_outarraym, 0), "out dev pointer");

		for (int ii = 0; ii < fullsize; ii++) {
				h_inarraym[ii].x = arrdis(arreng);
				h_inarraym[ii].y = arrdis(arreng);
		}

		PUSH_RANGE("Multi mapped FFT init", 1)
		hipfftHandle multiplan;
		geterror(hipfftPlanMany(&multiplan, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize), "mapped plan make");
		POP_RANGE

		PUSH_RANGE("Multi mapped FFT exec", 2)
		geterror(hipfftExecC2C(multiplan, d_inarray, d_inarray, HIPFFT_FORWARD), "mapped execution");
		poweraddkof<<<nblocks, nthreads>>>(d_inarray, d_outarray, fullsize / timesamp);
		geterror(hipGetLastError(), "mapped kernel exec");
		POP_RANGE

		geterror(hipfftDestroy(multiplan), "mapped plan destroy");
		geterror(hipHostFree(h_inarraym), "host in free");
		geterror(hipHostFree(h_outarraym), "host out free");

	} else if (mode == "a") {

		cout << "Will use asynchronous memory copies...\n";

	} else {
		cout << "Invalid memory mode option!! Will now quit!!";
	}

	geterror(hipfftDestroy(preinit), "init plan destroy");
	delete [] h_inarray;
	delete [] h_outarray;

    hipDeviceReset();

    return 0;

}

void printhelp(void)
{
	cout << "Test code for PAF FFT code" << endl << endl;
	cout << "Available options:" << endl;
	cout << "\t-p - switch pre-initialisation off" << endl;
	cout << "\t-t - use Thrust functions instead of custom kernels for power and averaging" << endl;
	cout << "\t-m - memory mode: n (default) - use hipMemcpy()" << endl;
	cout << "\t\tp - use pinned memory, m - use mapped pinned memory, a - use asynchronous copies" << endl;

}
