#include "hip/hip_runtime.h"
#include <algorithm>
#include <atomic>
#include <bitset>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <fstream>
#include <memory>
#include <sstream>
#include <thread>
#include <utility>
#include <vector>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <numa.h>
#include <pthread.h>

#include "config.hpp"
#include "dedisp/dedisp.hpp"
#include "dedisp/DedispPlan.hpp"
#include "errors.hpp"
#include "filterbank.hpp"
#include "filterbank_buffer.cuh"
#include "get_mjd.hpp"
#include "gpu_pool.cuh"
#include "heimdall/pipeline.hpp"
#include "kernels.cuh"
#include "obs_time.hpp"
#include "pdif.hpp"
#include "print_safe.hpp"

#include <inttypes.h>
#include <errno.h>
#include <netdb.h>
#include <arpa/inet.h>
#include <netinet/in.h>
#include <sys/mman.h>
#include <sys/socket.h>
#include <sys/wait.h>
#include <unistd.h>
#include <signal.h>

using std::atomic;
using std::cerr;
using std::endl;
using std::mutex;
using std::pair;
using std::string;
using std::thread;
using std::unique_ptr;
using std::vector;

bool GpuPool::working_ = true;

#define NFPGAS 48
#define NACCUMULATE 128

GpuPool::GpuPool(int poolid, InConfig config) : accumulate_(config.accumulate),
                                        avgfreq_(config.freqavg),
                                        avgtime_(config.timeavg),
                                        beamno_(0),
                                        codiflen_(config.codiflen),
                                        config_(config),
                                        dedispgulpsamples_(config.gulp),
                                        fftbatchsize_(config.nopols * config.nochans * config.accumulate * 128 / config.fftsize),
                                        fftedsize_(config.nopols * config.nochans * config.accumulate * 128 / config.fftsize * config.fftsize),
                                        fftpoints_(config.fftsize),
                                        filbits_(config.outbits),
                                        filchans_(config.filchans),
                                        gpuid_(config.gpuids[poolid]),
                                        gulpssent_(0),
                                        headlen_(config.headlen),
                                        ipstring_(config.ips[poolid]),
                                        // NOTE: There are config.nochans * config.accumulate * 128 8-byte words
                                        inbuffsize_(8  * config.nochans * config.accumulate * 128),
                                        inchans_(config.nochans),
                                        nopols_(config.nopols),
                                        noports_(config.noports),
                                        nostokes_(config.nostokes),
                                        nostreams_(config.nostreams),
                                        poolid_(poolid),
                                        ports_(config.ports),
                                        // NOTE: Quick hack to switch the scaling off
                                        scaled_(true),
                                        secondstorecord_(config.record),
                                        unpackedbuffersize_(config.nopols * config.nochans * config.accumulate * 128),
                                        verbose_(config.verbose) {

    start_ = std::chrono::system_clock::now();
    cores_ = thread::hardware_concurrency();
    // NOTE: Divide by 2 to get cores per Pool
    cores_ /= 2;
    cout << "Number of cores: " << cores_ << endl;
    if (cores_ == 0) {
        cerr << "Could not obtain the number of cores on node " << poolid << "!\n";
        cerr << "Will set to 10!" << endl;
        // NOTE: That should be 10 for the Effelsberg PAF machines - need to be careful when used on different machines.
        cores_ = 10;
    }

    if (verbose_)
        PrintSafe("Starting GPU pool", gpuid_);
}

void GpuPool::Initialise(void) {

    struct bitmask *mask = numa_parse_nodestring((std::to_string(poolid_)).c_str());
    numa_bind(mask);

    signal(SIGINT, GpuPool::HandleSignal);
    cudaCheckError(hipSetDevice(poolid_));

    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    CPU_SET((int)(poolid_) * cores_, &cpuset);
    int retaff = pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuset);

    if (retaff != 0) {
        PrintSafe("Error setting thread affinity for the GPU pool", poolid_);
        exit(EXIT_FAILURE);     // affinity is critical for us
    }

    // NOTE: The output number of channels has to be divisible by 4
    // This is a requirement for the dedisp/Heimdall GPU memory access
    // In this case, any power of 2, greater than 4 works
    // TODO: Test whether there can be a better way of doing this
    // Using the closest lower power of 2 can lose us a lot of channels
    // filchans_ = 1 << (int)log2f(filchans_);

    if (verbose_)
        PrintSafe("GPU pool for device", gpuid_, "running on CPU", sched_getcpu());

    // STAGE: PREPARE THE READ AND FILTERBANK BUFFERS
    if (verbose_)
        PrintSafe("Preparing the memory on pool", poolid_, "...");

    // Start of the page-locked memory allocation

    framenumbers_ = new int[accumulate_ * nostreams_];
    if (mlock(framenumbers_, accumulate_ * nostreams_ * sizeof(int))) {
        PrintSafe("Error on framenumbers_ mlock:", errno);
    }
    std::fill(framenumbers_, framenumbers_ + accumulate_ * nostreams_, -1);

    fpgaready_ = new atomic<long long>[accumulate_ * nostreams_];
    if (mlock(fpgaready_, accumulate_ * nostreams_ * sizeof(atomic<long long>))) {
        PrintSafe("Error on fpgaready_ mlock:", errno);
    }
    for (int isamp = 0; isamp < accumulate_ * nostreams_; ++isamp) {
        fpgaready_[isamp].store(0LL);
    }

    // NOTE: Each stream will have its own incoming buffer to read from
    hinbuffer_ = new unsigned char[inbuffsize_ * nostreams_];
    if (mlock(hinbuffer_, inbuffsize_ * nostreams_ * sizeof(unsigned char))) {
        PrintSafe("Error on hinbuffer_ mlock:", errno);
    }
    std::fill(hinbuffer_, hinbuffer_ + inbuffsize_ * nostreams_, 0);

    readybuffidx_ = new bool[NFPGAS * accumulate_ * nostreams_];
    if (mlock(readybuffidx_, (NFPGAS * accumulate_ * nostreams_) * sizeof(bool))) {
        PrintSafe("Error on readybuffidx_ mlock:", errno);
    }
    std::fill(readybuffidx_, readybuffidx_ + NFPGAS * accumulate_ * nostreams_, 0);

    receivebuffers_ = new unsigned char*[noports_];
    if (mlock(receivebuffers_, noports_ * sizeof(unsigned char*))) {
        PrintSafe("Error on receivebuffers_ mlock:", errno);
    }
    for (int iport = 0; iport < noports_; iport++) {
        receivebuffers_[iport] = new unsigned char[codiflen_ + headlen_];
        if (mlock(receivebuffers_[iport], (codiflen_ + headlen_) * sizeof(unsigned char))) {
            PrintSafe("Error on receivebuffers_ mlock for port", iport, ":", errno);
        }
    }

    // End of the page-locked memory allocation


    dedispplan_ = unique_ptr<DedispPlan>(new DedispPlan(filchans_, config_.tsamp, config_.ftop, config_.foff, gpuid_));
    filbuffer_ = unique_ptr<FilterbankBuffer>(new FilterbankBuffer(gpuid_));
    gpustreams_ = new hipStream_t[nostreams_];
    fftplans_ = new hipfftHandle[nostreams_];

    cudaCheckError(hipHostAlloc((void**)&hstreambuffer_, inbuffsize_ * nostreams_ * sizeof(unsigned char), hipHostMallocDefault));
    cudaCheckError(hipMalloc((void**)&dstreambuffer_, inbuffsize_ * nostreams_ * sizeof(unsigned char)));
    cudaCheckError(hipMalloc((void**)&dunpackedbuffer_, unpackedbuffersize_ * nostreams_ * sizeof(hipfftComplex)));
    cudaCheckError(hipMalloc((void**)&dfftedbuffer_, fftedsize_ * nostreams_ * sizeof(hipfftComplex)));

    hmeans_ = new float*[nostokes_];
    hrstdevs_ = new float*[nostokes_];

    // NOTE: Need to be careful what to fill the starting values with
    // We want to have the original data after the scaling in the fist run
    // So we can actually obtain the first scaling factors
    for (int istoke = 0; istoke < nostokes_; istoke++) {
        cudaCheckError(hipMalloc((void**)&hmeans_[istoke], filchans_ * sizeof(float)));
        cudaCheckError(hipMalloc((void**)&hrstdevs_[istoke], filchans_ * sizeof(float)));
    }

    cudaCheckError(hipMalloc((void**)&dmeans_, nostokes_ * sizeof(float*)));
    cudaCheckError(hipMalloc((void**)&drstdevs_, nostokes_ * sizeof(float*)));
    cudaCheckError(hipMemcpy(dmeans_, hmeans_, nostokes_ * sizeof(float*), hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(drstdevs_, hrstdevs_, nostokes_ * sizeof(float*), hipMemcpyHostToDevice));

    InitScaleFactors<<<1,filchans_,0,0>>>(dmeans_, drstdevs_, nostokes_);
    cudaCheckError(hipDeviceSynchronize());
    cudaCheckError(hipGetLastError());

    // STAGE: PREPARE THE DEDISPERSION
    // NOTE: generate_dm_list(dm_start, dm_end, width, tol)
    // width is the expected pulse width in microseconds
    // tol is the smearing tolerance factor between two DM trials
    dedispplan_->generate_dm_list(config_.dmstart, config_.dmend, 64.0f, 1.10f);


    /**
     * NOTE [Ewan]: We have hardcoded the extra portion of the buffer
     * to zero during debugging for Effelsberg. There is no reason to
     * believe that this can't be uncommented now, but I am leaving it
     * out as we are confident that the system is working right now.
     */
    //dedispextrasamples_ = dedispplan_->get_max_delay();
    dedispextrasamples_ = 0;
    dedispdispersedsamples_ = (size_t)dedispgulpsamples_ + dedispextrasamples_;
    //dedispnobuffers_ = (dedispdispersedsamples_ - 1) / dedispgulpsamples_ + 1;

    /**
     * Note [Ewan]: Same sentiment as above. This is commented out for debugging, but
     * can likely be renabled safely.
     */
    dedispnobuffers_  = 2;
    dedispbuffersize_ = dedispnobuffers_ * dedispgulpsamples_ + dedispextrasamples_;
    filbuffer_->Allocate(accumulate_, dedispnobuffers_, dedispextrasamples_, dedispgulpsamples_, dedispbuffersize_, filchans_, nostokes_, filbits_);
    dedispplan_->set_killmask(&config_.killmask[0]);

    // STAGE: PREPARE THE SINGLE PULSE SEARCH
    if (verbose_)
        PrintSafe("Setting up dedispersion and single pulse search on pool", poolid_, "...");

    SetSearchParams(singleparams_, config_);
    // NOTE: Commented out for the filterbank dump mode
    //hd_create_pipeline(&pipeline, params);
    // NOTE: Everything should be ready for single pulse search after this point

    // STAGE: start processing
    // FFT threads

    // NOTE: It has to be an array and I can't do anything about that
    fftsizes_[0] = (int)fftpoints_;

    for (int igstream = 0; igstream < nostreams_; igstream++) {
            cudaCheckError(hipStreamCreate(&gpustreams_[igstream]));
            cufftCheckError(hipfftPlanMany(&fftplans_[igstream], 1, fftsizes_, NULL, 1, fftpoints_, NULL, 1, fftpoints_, HIPFFT_C2C, fftbatchsize_));
            cufftCheckError(hipfftSetStream(fftplans_[igstream], gpustreams_[igstream]));
            gputhreads_.push_back(thread(&GpuPool::FilterbankData, this, igstream));
    }

    cudaCheckError(hipStreamCreate(&dedispstream_));
    gputhreads_.push_back(thread(&GpuPool::SendForDedispersion, this));

    // STAGE: Networking
    if (verbose_)
        PrintSafe("Setting up networking on pool", poolid_, "...");

    int netrv;
    addrinfo hints, *servinfo, *tryme;
    memset(&hints, 0, sizeof(hints));
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_DGRAM;
    hints.ai_flags = AI_PASSIVE;

    filedesc_ = new int[noports_];

    std::ostringstream oss;
    std::string strport;

    memset(&starttime_, sizeof(starttime_), 0);
    starttime_.refframe = -1;

    someonechecking_.store(false);

    // all the magic happens here
    for (int iport = 0; iport < noports_; iport++) {
        // TODO: Read port numbers from the config file
        oss.str("");
        oss << ports_.at(iport);
        strport = oss.str();

        if ((netrv = getaddrinfo(ipstring_.c_str(), strport.c_str(), &hints, &servinfo)) != 0)
            PrintSafe("getaddrinfo() error:", gai_strerror(netrv), "on pool", poolid_);

        for (tryme = servinfo; tryme != NULL; tryme=tryme->ai_next) {
            if((filedesc_[iport] = socket(tryme->ai_family, tryme->ai_socktype, tryme->ai_protocol)) == -1) {
                PrintSafe("Socket error on pool", poolid_);
                continue;
            }

            if (bind(filedesc_[iport], tryme->ai_addr, tryme->ai_addrlen) == -1) {
                close(filedesc_[iport]);
                PrintSafe("Bind error on pool", poolid_);
                continue;
            }
            break;
        }

        if (tryme == NULL)
            PrintSafe("Failed to bind to the socket", ports_.at(iport), "on pool", poolid_);
    }

    for (int iport = 0; iport < noports_; iport++)
        receivethreads_.push_back(thread(&GpuPool::ReceiveData, this, iport, ports_.at(iport)));

}

GpuPool::~GpuPool(void) {

    for(int ithread = 0; ithread < gputhreads_.size(); ithread++)
        gputhreads_[ithread].join();

    for (int ithread = 0; ithread < noports_; ithread++)
        receivethreads_[ithread].join();

    stop_ = std::chrono::system_clock::now();

    std::chrono::duration<double> diff = stop_ - start_;

    cout << "Pipeline execution time: " << std::chrono::duration_cast<std::chrono::seconds>(diff).count() << "s" << endl;

    // NOTE: Save the scaling factors before quitting
    if (scaled_) {
        string scalename = config_.outdir + "/scale_beam_" + std::to_string(beamno_) + ".dat";
        std::fstream scalefile(scalename.c_str(), std::ios_base::out | std::ios_base::trunc);

        if (scalefile) {
            float *means = new float[filchans_];
            float *stdevs = new float[filchans_];
            for (int istoke = 0; istoke < nostokes_; istoke++) {
                cudaCheckError(hipMemcpy(means, hmeans_[istoke], filchans_ * sizeof(float), hipMemcpyDeviceToHost));
                cudaCheckError(hipMemcpy(stdevs, hrstdevs_[istoke], filchans_ * sizeof(float), hipMemcpyDeviceToHost));
                for (int jj = 0; jj < filchans_; jj++) {
                    scalefile << means[jj] << " " << stdevs[jj] << endl;
                }
                scalefile << endl << endl;
            }
        }
        scalefile.close();
    }

    // NOTE: The filterbank buffer has to be deallocated separately
    filbuffer_->Deallocate();
    delete [] framenumbers_;
    delete [] gpustreams_;
    delete [] hinbuffer_;
    delete [] readybuffidx_;
    delete [] filedesc_;
    for (int iport = 0; iport < noports_; iport++) {
        delete [] receivebuffers_[iport];
    }
    delete [] receivebuffers_;


    for (int istoke = 0; istoke < nostokes_; istoke++) {
        cudaCheckError(hipFree(hmeans_[istoke]));
        cudaCheckError(hipFree(hrstdevs_[istoke]));
    }

    delete [] hmeans_;
    delete [] hrstdevs_;
    cudaCheckError(hipFree(dmeans_));
    cudaCheckError(hipFree(drstdevs_));

    cudaCheckError(hipFree(dstreambuffer_));
    cudaCheckError(hipFree(dfftedbuffer_));
    cudaCheckError(hipHostFree(hstreambuffer_));

    for (int igstream = 0; igstream < nostreams_; igstream++) {
        cufftCheckError(hipfftDestroy(fftplans_[igstream]));
    }

    delete [] fftplans_;
}

void GpuPool::HandleSignal(int signum) {
    PrintSafe("Captured the signal!\nWill now terminate!\n");
    working_ = false;
}

void GpuPool::FilterbankData(int stream) {

    cudaCheckError(hipSetDevice(gpuid_));
    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    CPU_SET((int)(poolid_) * cores_ + 1 + (int)(stream / 1), &cpuset);
    int retaff = pthread_setaffinity_np(gputhreads_[stream].native_handle(), sizeof(cpu_set_t), &cpuset);

    if (retaff != 0) {
        PrintSafe("Error setting thread affinity for stream", stream, "on pool", poolid_);
        exit(EXIT_FAILURE);
    }

    if (verbose_)
        PrintSafe("Starting worker", stream, "on pool", poolid_, "on CPU", sched_getcpu());

    unsigned int skip = stream * unpackedbuffersize_;
    unsigned char **pfil = filbuffer_ -> GetFilPointer();

    pair<unsigned char*, int> bufferinfo;
    unsigned char* incoming;
    ObsTime incomingtime = {0, 0, 0};

    while (working_) {
        // NOTE: Time this portion of the code has to be profiled carefully as unique_lock can be a bit expensive
        std::unique_lock<mutex> worklock(workmutex_);
        workready_.wait(worklock, [this]{return (!workqueue_.empty() || !working_);});
        if (working_) {
            // TODO: Copy the data using the information in the queue
            bufferinfo = workqueue_.front();
            workqueue_.pop();
            worklock.unlock();

            // NOTE: This already has the correct offset for a given buffer chunk included
            incoming = bufferinfo.first;
            incomingtime.refframe = bufferinfo.second;
            // TODO: Check whether we actually need this intermediate buffer or could we just copy directly to the GPU
            std::copy(incoming, incoming + inbuffsize_, hstreambuffer_ + stream * inbuffsize_);

            incomingtime.refepoch = starttime_.refepoch;
            incomingtime.refsecond = starttime_.refsecond;
            cudaCheckError(hipMemcpyAsync(dstreambuffer_ + stream * inbuffsize_, hstreambuffer_ + stream * inbuffsize_, inbuffsize_, hipMemcpyHostToDevice, gpustreams_[stream]));
            UnpackKernel<<<48, 128, 0, gpustreams_[stream]>>>(reinterpret_cast<int2*>(dstreambuffer_ + stream * inbuffsize_), dunpackedbuffer_ + skip);
            cufftCheckError(hipfftExecC2C(fftplans_[stream], dunpackedbuffer_ + skip, dfftedbuffer_ + skip, HIPFFT_FORWARD));
            DetectScrunchKernel<<<2 * NACCUMULATE, 1024, 0, gpustreams_[stream]>>>(dfftedbuffer_ + skip, reinterpret_cast<float*>(pfil[0]), filchans_, dedispnobuffers_, dedispgulpsamples_, dedispextrasamples_, incomingtime.refframe);
            hipStreamSynchronize(gpustreams_[stream]);
            cudaCheckError(hipGetLastError());
            filbuffer_ -> UpdateFilledTimes(incomingtime);
        }
    }
}

void GpuPool::SendForDedispersion(void) {

    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);

    //Note [Ewan]: multiply by 10 to match pacifix numa layout (should not be hardcoded)
    CPU_SET((int)(poolid_) * cores_ , &cpuset);
    int retaff = pthread_setaffinity_np(gputhreads_[nostreams_].native_handle(), sizeof(cpu_set_t), &cpuset);
    if (retaff != 0) {
        PrintSafe("Error setting thread affinity for dedisp thread on pool", poolid_);
        exit(EXIT_FAILURE);
    }

    {
        std::unique_lock<std::mutex> framelock(framemutex_);
        startrecord_.wait(framelock, [this]{return starttime_.refframe != -1;});
    }

    ObsTime sendtime;

    header_f headerfil;
    headerfil.raw_file = "tastytastytest";
    headerfil.source_name = "J1641-45";
    headerfil.fch1 = config_.ftop;
    // NOTE: For channels in decreasing order
    headerfil.foff = -1.0 * abs(config_.foff);
    headerfil.rdm = 0.0;
    headerfil.tsamp = config_.tsamp;
    headerfil.data_type = 1;
    headerfil.ibeam = beamno_;
    headerfil.machine_id = 2;
    headerfil.nbeams = 1;
    headerfil.nbits = filbits_;
    headerfil.nchans = filchans_;
    headerfil.nifs = 1;
    headerfil.telescope_id = 8;

    cout << "Filled the header info " << beamno_ << endl;

    cudaCheckError(hipSetDevice(gpuid_));
    if (verbose_)
        PrintSafe("Dedisp thread up and running on pool", poolid_, "...");

    int ready{0};
    while(working_) {
        ready = filbuffer_->CheckIfReady();
        if (ready) {
            if (scaled_) {
                // TODO: Will we be able to update this information during the observation?
                headerfil.az = 0.0;
                headerfil.za = 0.0;
                headerfil.ra = config_.ra;
                headerfil.dec = config_.dec;
                // TODO: This totally doesn't work when something is skipped
                // Need to move to the version that uses the frame number of the chunk being sent
                headerfil.tstart = GetMjd(starttime_.refepoch, starttime_.refsecond + 27 + (gulpssent_ + 1)* dedispgulpsamples_ * config_.tsamp);
                sendtime = filbuffer_->GetTime(ready-1);
                //headerfil.tstart = GetMjd(sendtime.startepoch, sendtime.startsecond + 27 + sendtime.framefromstart * config_.tsamp);
                // TODO: This line doesn't work - fix this! Possible bug related to multiple time samples per frame

                if (verbose_)
                    PrintSafe(ready - 1, "buffer ready on pool", poolid_);

                filbuffer_ -> SendToRam(ready, dedispstream_, (gulpssent_ % 2));
                filbuffer_ -> SendToDisk((gulpssent_ % 2), headerfil, config_.outdir);
                // TODO: Possible race condition
                gulpssent_++;

                if (verbose_)
                    PrintSafe("Filterbank", gulpssent_, "with MJD", headerfil.tstart, "for beam", beamno_, "on pool", poolid_, "saved");

                // NOTE: This fails from time to time and pipeline finishes much earlier than expected
                // TODO: Fix it!
                if ((int)(gulpssent_ * dedispdispersedsamples_ * config_.tsamp) >= secondstorecord_)
                    working_ = false;

            }   else {
                // perform the scaling
                // NOTE: Scaling breaks down when there is no data - division by a standard deviation of 0
                // TODO: Need to come up with a more clever way of dealing with that
                // filbuffer_->GetScaling(ready, dedispstream_, dmeans_, drstdevs_);
                cudaCheckError(hipGetLastError());
                scaled_ = true;
                ready = 0;

                if (verbose_)
                    PrintSafe("Scaling factors have been obtained on pool", poolid_);
            }

        } else {
            std::this_thread::yield();
        }
    }
}

void GpuPool::ReceiveData(int portid, int recport) {
    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    // NOTE: 2 ports per CPU core
    CPU_SET((int)(poolid_) * cores_ + 1 + nostreams_ + (int)(portid / 2), &cpuset);
    int retaff = pthread_setaffinity_np(receivethreads_[portid].native_handle(), sizeof(cpu_set_t), &cpuset);
    if (retaff != 0) {
        PrintSafe("Error setting thread affinity for receive thread on port", recport, "on pool", poolid_);
        exit(EXIT_FAILURE);
    }

    if (verbose_)
        PrintSafe("Receive thread on port", recport, "on pool", poolid_, "running on CPU", sched_getcpu());

    sockaddr_storage senderaddr;
    memset(&senderaddr, 0, sizeof(senderaddr));
    socklen_t addrlen;
    memset(&addrlen, 0, sizeof(addrlen));
    addrlen = sizeof(senderaddr);

    int numbytes{0};
    short fpga{0};
    short bufidx{0};

    int frame{0};
    int modframe{0};
    int refsecond{0};
    int refframe{0};

    bool checkexpected;

    while (std::chrono::high_resolution_clock::now() < config_.recordstart) {
        if ((numbytes = recvfrom(filedesc_[portid], receivebuffers_[portid], codiflen_ + headlen_ - 1, 0, (struct sockaddr*)&senderaddr, &addrlen)) == -1)
            PrintSafe("recvfrom error on port", recport, "on pool", poolid_, "with code", errno);
    }

    if (portid == 0) {
        std::lock_guard<std::mutex> frameguard(framemutex_);
        unsigned char *tmpbuffer = receivebuffers_[0];
        numbytes = recvfrom(filedesc_[portid], receivebuffers_[portid], codiflen_ + headlen_ - 1, 0, (struct sockaddr*)&senderaddr, &addrlen);
        starttime_.refepoch = (int)(tmpbuffer[12] >> 2);
        starttime_.refsecond = (int)(tmpbuffer[3] | (tmpbuffer[2] << 8) | (tmpbuffer[1] << 16) | ((tmpbuffer[0] & 0x3f) << 24));
        starttime_.refframe = (int)(tmpbuffer[7] | (tmpbuffer[6] << 8) | (tmpbuffer[5] << 16) | (tmpbuffer[4] << 24));
        beamno_ = (int)(tmpbuffer[23] | (tmpbuffer[22] << 8));
        startrecord_.notify_all();
    } else {
        std::unique_lock<std::mutex> framelock(framemutex_);
        startrecord_.wait(framelock, [this]{return starttime_.refframe != -1;});
    }

    while(working_) {
        if ((numbytes = recvfrom(filedesc_[portid], receivebuffers_[portid], codiflen_ + headlen_ - 1, 0, (struct sockaddr*)&senderaddr, &addrlen)) == -1)
            PrintSafe("recvfrom error on port", recport, "on pool", poolid_, "with code", errno);

        if (numbytes == 0)
            continue;
        refsecond = (int)(receivebuffers_[portid][3] | (receivebuffers_[portid][2] << 8) | (receivebuffers_[portid][1] << 16) | ((receivebuffers_[portid][0] & 0x3f) << 24));
        frame = (int)(receivebuffers_[portid][7] | (receivebuffers_[portid][6] << 8) | (receivebuffers_[portid][5] << 16) | (receivebuffers_[portid][4] << 24));
        frame = frame + (refsecond - starttime_.refsecond) / 27 * 250000 - starttime_.refframe;
        fpga = ((short)((((struct sockaddr_in*)&senderaddr)->sin_addr.s_addr >> 16) & 0xff) - 1) * 6 + ((int)((((struct sockaddr_in*)&senderaddr)->sin_addr.s_addr >> 24)& 0xff) - 1) / 2;
        // NOTE: If we get a late frame coming in, it can have an absolute number less than 0
        // This will happen only at the beginning of receiving and should be unnecesary after first few packets
        if (frame < 0) {
            continue;
        }

        // NOTE: Which stream buffer the data is saved to
        bufidx = (int)(frame / accumulate_) % nostreams_;
        // NOTE: Number of packets to skip to get to the start of the stream buffer
        bufidx *= NFPGAS * NACCUMULATE;
        // NOTE: Correct FPGA within the stream buffer
        bufidx += fpga * NACCUMULATE;
        // NOTE: Correct frame packet within the stream buffer
        bufidx += (frame % accumulate_);

        modframe = frame % (accumulate_ * nostreams_);

        framenumbers_[modframe] = frame;
        std::copy(receivebuffers_[portid] + headlen_, receivebuffers_[portid] + codiflen_ + headlen_, hinbuffer_ + codiflen_ * bufidx);
        readybuffidx_[bufidx] = true;
        fpgaready_[modframe] |= (1LL << fpga);

        checkexpected = false;

        if (someonechecking_.compare_exchange_strong(checkexpected,true)) {
            // NOTE: Check the last sample of the current stream and something inside of the next
            for (int istream = 0; istream < nostreams_; ++istream) {
                // NOTE: Checking for at least 24 FPGAS - doesn't make much sense to be processing with less than half of the band
                // TODO: Make this a more strict constraint when FPGA problems are sorted out - a quarter or a third
                // NOTE: Check in the quarter of next stream - should give enough time for latecomers
                // NOTE: This part is not overly atomic - the value can be changed when it is being checked
                // TODO: Is it going to be much of a problem?
                if ((__builtin_popcountll(fpgaready_[(istream + 1) * accumulate_ - 1]) >= 24) && (__builtin_popcountll(fpgaready_[((istream + 1) % nostreams_) * accumulate_ + accumulate_ / 4]) >= 24)) {
                    for (int isamp = 0; isamp < accumulate_; ++isamp) {
                        fpgaready_[istream * accumulate_ + isamp].store(0LL);
                    }
                    for (int frameidx = 0; frameidx < accumulate_; ++frameidx) {
                        if (framenumbers_[istream * accumulate_ + frameidx] != -1) {
                            refframe = framenumbers_[istream * accumulate_ + frameidx] - frameidx;
                        }
                    }
                    // TODO: Fill the frame numbers with -1
                    std::lock_guard<mutex> worklock(workmutex_);
                    // NOTE: Push data onto the worker queue
                    // TODO: Decide which data actually goes there - preferably a pair, but that can be a performance hit
                    workqueue_.push(std::make_pair(hinbuffer_ + istream * inbuffsize_, refframe));
                    workready_.notify_one();
                    break;
                }
            }
            someonechecking_.store(false);
        }
    }
    // NOTE: Wakes the consumer threads up to let them know their struggle is over
    workready_.notify_all();
}
