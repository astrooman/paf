#include "hip/hip_runtime.h"
#include <algorithm>
#include <bitset>
#include <iostream>
#include <fstream>
#include <memory>
#include <mutex>
#include <queue>
#include <sstream>
#include <thread>
#include <utility>
#include <vector>

#include <boost/array.hpp>
#include <boost/asio.hpp>
#include <boost/bind.hpp>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <numa.h>
#include <pthread.h>
#include <thrust/device_vector.h>

#include "buffer.cuh"
#include "config.hpp"
#include "dedisp/dedisp.hpp"
#include "dedisp/DedispPlan.hpp"
#include "errors.hpp"
#include "filterbank.hpp"
#include "get_mjd.hpp"
#include "heimdall/pipeline.hpp"
#include "kernels.cuh"
#include "paf_metadata.hpp"
#include "pdif.hpp"
#include "pool_multi.cuh"

#include <inttypes.h>
#include <errno.h>
#include <netdb.h>
#include <arpa/inet.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <sys/wait.h>
#include <unistd.h>
#include <signal.h>

using std::cout;
using std::endl;
using std::mutex;
using std::pair;
using std::queue;
using std::thread;
using std::unique_ptr;
using std::vector;

#define BYTES_PER_WORD 8
#define HEADER 64
#define WORDS_PER_PACKET 896
#define BUFLEN 7232
#define PORTS 8

mutex cout_guard;

/* ########################################################
TODO: Too many copies - could I use move in certain places?
#########################################################*/

/*##############################################
IMPORTANT: from what I seen in the system files:
eth3, gpu0, gpu1 - NUMA node 0, CPUs 0-7
eth2, gpu2, gpu3 - NUMA node 1, CPUs 8-15
##############################################*/

Oberpool::Oberpool(config_s config) : ngpus(config.ngpus)
{

    for (int ii = 0; ii < ngpus; ii++) {
        gpuvector.push_back(unique_ptr<GPUpool>(new GPUpool(ii, config)));
    }

    for (int ii = 0; ii < ngpus; ii++) {
        threadvector.push_back(thread(&GPUpool::execute, std::move(gpuvector[ii])));
    }

}

Oberpool::~Oberpool(void)
{
    for (int ii = 0; ii < ngpus; ii++) {
        threadvector[ii].join();
    }

}

bool GPUpool::working_ = true;

GPUpool::GPUpool(int id, config_s config) : accumulate(config.accumulate),
                                        beamno{0},
                                        gpuid(config.gpuids[id]),
                                        strip(config.ips[id]),
                                        highest_buf(0),
                                        batchsize(config.batch),
                                        fftpoint(config.fftsize),
                                        timeavg(config.timesavg),
                                        freqavg(config.freqavg),
                                        nostreams(config.streamno),
                                        npol(config.npol),
                                        poolid_(id),
                                        d_rearrange_size(8 * config.batch * config.fftsize * config.timesavg * config.accumulate),
                                        d_in_size(config.batch * config.fftsize * config.timesavg * config.npol * config.accumulate),
                                        d_fft_size(config.batch * config.fftsize * config.timesavg * config.npol * config.accumulate),
                                        d_time_scrunch_size((config.fftsize - 5) * config.batch * config.accumulate),
                                        d_freq_scrunch_size((config.fftsize - 5) * config.batch  * config.accumulate / config.freqavg),
                                        gulps_sent(0),
                                        gulps_processed(0),
                                        working(true),
					                    packcount(0),
                                        verbose_(config.verbose)

{
    avt = min(nostreams + 2, thread::hardware_concurrency());

    config_ = config;

    if (verbose_) {
        cout_guard.lock();
        cout << "Starting GPU pool " << gpuid << endl;
	    cout.flush();
        cout_guard.unlock();
    }
}

void GPUpool::execute(void)
{
    struct bitmask *mask = numa_parse_nodestring((std::to_string(poolid_)).c_str());
    numa_bind(mask);

    signal(SIGINT, GPUpool::HandleSignal);
    cudaCheckError(hipSetDevice(poolid_));

    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    CPU_SET((int)(poolid_) * 8, &cpuset);
    int retaff = pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuset);

    if (retaff != 0) {
        cout_guard.lock();
        cerr << "Error setting thread affinity for the GPU pool " << gpuid << endl;
        cout_guard.unlock();
    }

    if(verbose_) {
        cout_guard.lock();
        cout << "GPU pool for device " << gpuid << " running on CPU " << sched_getcpu() << endl;
        cout_guard.unlock();
    }

    p_dedisp = unique_ptr<DedispPlan>(new DedispPlan(config_.filchans, config_.tsamp, config_.ftop, config_.foff, gpuid));
    p_mainbuffer = unique_ptr<Buffer<float>>(new Buffer<float>(gpuid));

    frame_times = new int[accumulate * nostreams];
    // every thread will be associated with its own CUDA streams
    mystreams = new hipStream_t[avt];
    // each worker stream will have its own cuFFT plan
    myplans = new hipfftHandle[nostreams];

    int nkernels = 4;
    CUDAthreads = new unsigned int[nkernels];
    CUDAblocks = new unsigned int[nkernels];
    // TODO: make a private const data memmber and put in the initializer list!!
    nchans = config_.nchans;

    CUDAthreads[0] = 7;
    CUDAthreads[1] = fftpoint * timeavg * batchsize / 42;
    CUDAthreads[2] = nchans;		// 21 - fine!
    CUDAthreads[3] = batchsize * 27 / freqavg;	// 63 - fine!

    CUDAblocks[0] = 48;
    CUDAblocks[1] = 42;
    CUDAblocks[2] = 1;
    CUDAblocks[3] = 1;

    // STAGE: PREPARE THE READ AND FILTERBANK BUFFERS
    if (verbose_)
        cout << "Preparing the memory..." << endl;

    cdesc = hipCreateChannelDesc<int2>();
    cudaCheckError(hipPeekAtLastError());

    d_array2Dp = new hipArray*[nostreams];
    texObj = new hipTextureObject_t[nostreams];
    rdesc = new hipResourceDesc[nostreams];
    tdesc = new hipTextureDesc[nostreams];
    for (int ii = 0; ii < nostreams; ii++) {
        cudaCheckError(hipMallocArray(&(d_array2Dp[ii]), &cdesc, 7, (batchsize  / 7) * fftpoint * timeavg * accumulate));

        memset(&(rdesc[ii]), 0, sizeof(hipResourceDesc));
        rdesc[ii].resType = hipResourceTypeArray;
        rdesc[ii].res.array.array = d_array2Dp[ii];

        memset(&(tdesc[ii]), 0, sizeof(hipTextureDesc));
        tdesc[ii].addressMode[0] = hipAddressModeClamp;
        tdesc[ii].filterMode = hipFilterModePoint;
        tdesc[ii].readMode = hipReadModeElementType;

        texObj[ii] = 0;
        cudaCheckError(hipCreateTextureObject(&(texObj[ii]), &(rdesc[ii]), &(tdesc[ii]), NULL));
    }

    // it has to be an array and I can't do anything about that
    sizes[0] = (int)fftpoint;

    // each stream will have its own incoming buffeer to read from
    pack_per_buf = batchsize / 7 * accumulate * nostreams;
    h_pol = new unsigned char[d_rearrange_size * nostreams];
    bufidx_array = new bool[pack_per_buf]();
    cudaCheckError(hipHostAlloc((void**)&h_in, d_rearrange_size * nostreams * sizeof(unsigned char), hipHostMallocDefault));
    cudaCheckError(hipMalloc((void**)&d_in, d_in_size * nostreams * sizeof(hipfftComplex)));
    cudaCheckError(hipMalloc((void**)&d_fft, d_fft_size * nostreams * sizeof(hipfftComplex)));
    // need to store all 4 Stoke parameters
    dv_time_scrunch.resize(nostreams);
    dv_freq_scrunch.resize(nostreams);
    // TODO: make a private const data memmber and put in the initializer list!!
    stokes = config_.stokes;
    for (int ii = 0; ii < nostreams; ii++) {
        dv_time_scrunch[ii].resize(d_time_scrunch_size * stokes);
        dv_freq_scrunch[ii].resize(d_freq_scrunch_size * stokes);
    }

    // STAGE: PREPARE THE DEDISPERSION
    // generate_dm_list(dm_start, dm_end, width, tol)
    // width is the expected pulse width in microseconds
    // tol is the smearing tolerance factor between two DM trials
    p_dedisp->generate_dm_list(config_.dstart, config_.dend, 64.0f, 1.10f);
    // this is the number of time sample - each timesample will have config.filchans frequencies
    dedisp_totsamples = (size_t)config_.gulp + 1; //p_dedisp->get_max_delay();
    dedisp_buffno = (dedisp_totsamples - 1) / config_.gulp + 1;
    dedisp_buffsize = dedisp_buffno * config_.gulp + 1; //p_dedisp->get_max_delay();
    if (verbose_)
        cout << "Total buffer size: " << dedisp_buffsize << endl;
    // can this method be simplified?
    p_mainbuffer->allocate(accumulate, dedisp_buffno, 1, config_.gulp, dedisp_buffsize, config_.filchans, stokes);
    buffer_ready[0] = false;
    buffer_ready[1] = false;
    p_dedisp->set_killmask(&config_.killmask[0]);
    // everything should be ready for dedispersion after this point

    // STAGE: PREPARE THE SINGLE PULSE SEARCH
    if (verbose_)
        cout << "Setting up dedispersion and single pulse search..." << endl;
    set_search_params(params, config_);
    //commented out for the filterbank dump mode
    //hd_create_pipeline(&pipeline, params);
    // everything should be ready for single pulse search after this point

    // STAGE: start processing
    // FFT threads
    for (int ii = 0; ii < nostreams; ii++) {
            cudaCheckError(hipStreamCreate(&mystreams[ii]));
            // TODO: add separate error checking for cufft functions
            cufftCheckError(hipfftPlanMany(&myplans[ii], 1, sizes, NULL, 1, fftpoint, NULL, 1, fftpoint, HIPFFT_C2C, batchsize * timeavg * npol * accumulate));
            cufftCheckError(hipfftSetStream(myplans[ii], mystreams[ii]));
            mythreads.push_back(thread(&GPUpool::worker, this, ii));
    }

    // dedispersion thread
    cudaCheckError(hipStreamCreate(&mystreams[avt - 2]));
    mythreads.push_back(thread(&GPUpool::dedisp_thread, this, avt - 2));

    // STAGE: networking
    if (verbose_)
        cout << "Setting up networking..." << endl;

    memset(&start_time, 0, sizeof(start_time)) ;
    int netrv;
    addrinfo hints, *servinfo, *tryme;
    memset(&hints, 0, sizeof(hints));
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_DGRAM;
    hints.ai_flags = AI_PASSIVE;

    sfds = new int[PORTS];

    rec_bufs = new unsigned char*[PORTS];
    for (int ii = 0; ii < PORTS; ii++)
        rec_bufs[ii] = new unsigned char[BUFLEN];

    std::ostringstream oss;
    std::string strport;

    // all the magic happens here
    for (int ii = 0; ii < PORTS; ii++) {
        oss.str("");
        oss << 17100 + ii;
        strport = oss.str();

        if((netrv = getaddrinfo(strip.c_str(), strport.c_str(), &hints, &servinfo)) != 0) {
            cout_guard.lock();
            cerr <<  "getaddrinfo() error: " << gai_strerror(netrv) << endl;
            cout_guard.unlock();
        }

        for (tryme = servinfo; tryme != NULL; tryme=tryme->ai_next) {
            if((sfds[ii] = socket(tryme->ai_family, tryme->ai_socktype, tryme->ai_protocol)) == -1) {
                cout_guard.lock();
                cerr << "Socket error\n";
                cout_guard.unlock();
                continue;
            }

            if(bind(sfds[ii], tryme->ai_addr, tryme->ai_addrlen) == -1) {
                close(sfds[ii]);
                cout_guard.lock();
                cerr << "Bind error\n";
                cout_guard.unlock();
                continue;
            }
            break;
        }

        if (tryme == NULL) {
            cout_guard.lock();
            cerr << "Failed to bind to the socket " << 17100 + ii << "\n";
            cout_guard.unlock();
        }
    }

    int bufres{4*1024*1024};    // 4MB

    for (int ii = 0; ii < PORTS; ii++) {
        if(setsockopt(sfds[ii], SOL_SOCKET, SO_RCVBUF, (char *)&bufres, sizeof(bufres)) != 0) {
            cout_guard.lock();
            cerr << "Setsockopt error on port " << 17100 + ii << endl;
            cerr << "Errno " << errno << endl;
            cout_guard.unlock();
        }
    }

    for (int ii = 0; ii < PORTS; ii++)
        receive_threads.push_back(thread(&GPUpool::receive_thread, this, ii));

//    for (int ii = 0; ii < PORTS; ii++)
//        receive_threads[ii].join();

    // TODO: this thread does nothing at this point so might as well make it listen to metadata
    if (verbose_)
        cout << "Setting up metadata logger..." << endl;

    int metabytes, sock_meta;
    addrinfo hints_meta, *servinfo_meta, *tryme_meta;
    memset(&hints_meta, 0, sizeof(hints_meta));
    hints_meta.ai_family = AF_INET;
    hints_meta.ai_socktype = SOCK_DGRAM;
    hints_meta.ai_flags = AI_PASSIVE;
    sockaddr_storage meta_addr;
    memset(&meta_addr, 0, sizeof(meta_addr));
    socklen_t meta_len;

    if ((netrv = getaddrinfo(NULL, "26666", &hints_meta, &servinfo_meta)) != 0) {
        cout_guard.lock();
        cerr << "gettaddrinfo() error on metadata socket 26666" << endl;
        cout_guard.unlock();
    }

        for (tryme_meta = servinfo_meta; tryme_meta != NULL; tryme_meta=tryme_meta->ai_next) {
            if ((sock_meta = socket(tryme_meta->ai_family, tryme_meta->ai_socktype, tryme_meta->ai_protocol)) == -1) {
                cout_guard.lock();
                cerr << "Metadata socket error\n";
                cout_guard.unlock();
                continue;
            }
            if (bind(sock_meta, tryme_meta->ai_addr, tryme_meta->ai_addrlen) == -1) {
                cout_guard.lock();
                cerr << "Metadata bind error\n";
                cout_guard.unlock();
                continue;
            }
            break;
        }

    if (tryme_meta == NULL) {
        cout_guard.lock();
        cerr << "Failed to bind to the metadata socket\n";
        cout_guard.unlock();
    }

    metadata paf_meta;
    ostringstream ossmeta;
    ossmeta << "metadata_log_" << beamno << ".log";
    string metafile = config_.outdir + "/" + ossmeta.str();
    std::fstream metalog(metafile.c_str(), std::ios_base::out | std::ios_base::trunc);

    char *metabuffer = new char[4096];
    meta_len = sizeof(meta_addr);
    if (metalog) {
        /*while(working_) {
            metabytes = recvfrom(sock_meta, metabuffer, 4096, 0, (struct sockaddr*)&meta_addr, &meta_len);
            if (metabytes != 0) {
                string metastr(metabuffer);
                paf_meta.getMetaData(metastr, 0);
                cout << paf_meta.timestamp << "\t";
                cout << paf_meta.beam_num << "\t";
                cout << paf_meta.beam_ra << "\t";
                cout << paf_meta.beam_dec << "\t";
                cout << paf_meta.target_name << endl;
                cout.flush();

                metalog << paf_meta.timestamp << "\t";
                metalog << paf_meta.beam_num << "\t";
                metalog << paf_meta.beam_ra << "\t";
                metalog << paf_meta.beam_dec << "\t";
                metalog << paf_meta.target_name << endl << endl;
            } else {
                cerr << "Got nothing from metadata" << endl;
            }
        }
       */
        metalog.close();
    } else {
        cout_guard.lock();
        cerr << "Metadata log file error!!" << endl;
        cout_guard.unlock();
    }
    delete [] metabuffer;
}

GPUpool::~GPUpool(void)
{
    // TODO: clear the memory properly
    if (verbose_)
        cout << "Calling destructor" << endl;

    for(int ii = 0; ii < mythreads.size(); ii++)
        mythreads[ii].join();

    for (int ii = 0; ii < PORTS; ii++)
        receive_threads[ii].join();

    // cleaning up the stuff
    for (int ii = 0; ii < nostreams; ii++) {
        cudaCheckError(hipDestroyTextureObject(texObj[ii]));
        cudaCheckError(hipFreeArray(d_array2Dp[ii]));
    }

    // need deallocation in the dedisp buffer destructor as well
    p_mainbuffer->deallocate();
    // this stuff is deleted in order it appears in the code
    delete [] frame_times;
    delete [] mystreams;
    delete [] CUDAthreads;
    delete [] CUDAblocks;
    delete [] d_array2Dp;
    delete [] texObj;
    delete [] rdesc;
    delete [] tdesc;
    delete [] h_pol;
    delete [] bufidx_array;
    delete [] sfds;
    for (int ii = 0; ii < PORTS; ii++) {
        delete [] rec_bufs[ii];
    }
    delete [] rec_bufs;

    cudaCheckError(hipFree(d_in));
    cudaCheckError(hipFree(d_fft));
    cudaCheckError(hipHostFree(h_in));

    for (int ii = 0; ii < nostreams; ii++) {
        cufftCheckError(hipfftDestroy(myplans[ii]));
    }

    delete [] myplans;
}

void GPUpool::HandleSignal(int signum) {

    if (verbose_)
        cout << "Captured the signal\nWill now terminate!\n";
    working_ = false;
}

void GPUpool::worker(int stream)
{

    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    CPU_SET((int)(poolid_) * 8 + 1 + (int)(stream / 1), &cpuset);
    int retaff = pthread_setaffinity_np(mythreads[stream].native_handle(), sizeof(cpu_set_t), &cpuset);

    if (retaff != 0) {
        cout_guard.lock();
        cerr << "Error setting thread affinity for stream " << stream << endl;
        cout_guard.unlock();
    }

    if (verbose_) {
        cout_guard.lock();
        cout << "Starting worker " << gpuid << ":" << stream << " on CPU " << sched_getcpu() << endl;
        cout_guard.unlock();
    }

    hipSetDevice(gpuid);
    dim3 rearrange_b(1,48,1);
    dim3 rearrange_t(7,1,1);
    unsigned int skip = stream * d_in_size;

    unsigned int current_frame;

    float *pdv_time_scrunch = thrust::raw_pointer_cast(dv_time_scrunch[stream].data());
    float *pdv_freq_scrunch = thrust::raw_pointer_cast(dv_freq_scrunch[stream].data());

    float **p_fil = p_mainbuffer->get_pfil();
    float **pd_fil;
    hipMalloc((void**)&pd_fil, stokes * sizeof(float *));
    hipMemcpy(pd_fil, p_fil, stokes * sizeof(float *), hipMemcpyHostToDevice);

    int skip_read = stream * (pack_per_buf / nostreams);
    int skip_to_end = (stream + 1) * (pack_per_buf / nostreams) - 1;
    int next_start;
    if (stream != 3) {
        next_start = skip_to_end + 24;
    } else {
        next_start = 23;
    }
    bool endready = false;
    bool innext = false;
    while (working_) {
        endready = false;
        innext = false;
        for (int ii = 0; ii < 4; ii++) {
            //cout << bufidx_array[skip_to_end - ii] << " " << bufidx_array[skip_to_end + 24 - ii] << endl;
            endready = endready || bufidx_array[skip_to_end - ii];
            innext = innext || bufidx_array[next_start - ii];
        }
        if (endready && innext) {
            for (int ii = 0; ii < 4; ii++) {
                bufidx_array[skip_to_end - ii] = false;
                bufidx_array[next_start - ii] = false;
            }
            std::copy(h_pol + stream * d_rearrange_size,  h_pol + stream * d_rearrange_size + d_rearrange_size, h_in + stream * d_rearrange_size);;
            for (int frameidx = 0; frameidx < accumulate; frameidx++) {
                if (frame_times[stream * accumulate + frameidx] != 0) {
                    current_frame = frame_times[stream * accumulate + frameidx];
                    break;
                }
            }
            for (int frameidx = 0; frameidx < accumulate; frameidx++)
                frame_times[stream * accumulate + frameidx] = 0;

            obs_time frame_time{start_time.start_epoch, start_time.start_second, current_frame};
            cudaCheckError(cudaMemcpyToArrayAsync(d_array2Dp[stream], 0, 0, h_in + stream * d_rearrange_size, d_rearrange_size, hipMemcpyHostToDevice, mystreams[stream]));
            rearrange2<<<rearrange_b, rearrange_t, 0, mystreams[stream]>>>(texObj[stream], d_in + skip, accumulate);
            cufftCheckError(hipfftExecC2C(myplans[stream], d_in + skip, d_fft + skip, HIPFFT_FORWARD));
            powertime2<<<48, 27, 0, mystreams[stream]>>>(d_fft + skip, pdv_time_scrunch, d_time_scrunch_size, timeavg, accumulate);
            addchannel2<<<CUDAblocks[3], CUDAthreads[3], 0, mystreams[stream]>>>(pdv_time_scrunch, pd_fil, (short)config_.filchans, config_.gulp, dedisp_buffsize, dedisp_buffno, d_time_scrunch_size, freqavg, current_frame, accumulate);
            hipStreamSynchronize(mystreams[stream]);
            // used to check for any possible errors in the kernel execution
            cudaCheckError(hipGetLastError());
            //cout << current_frame << endl;
            //cout.flush();
            p_mainbuffer->update(frame_time);
            //working_ = false;
        } else {
            std::this_thread::yield();
        }
    }

    hipFree(pd_fil);
}

void GPUpool::dedisp_thread(int dstream)
{

    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    CPU_SET((int)(poolid_) * 8, &cpuset);
    int retaff = pthread_setaffinity_np(mythreads[nostreams].native_handle(), sizeof(cpu_set_t), &cpuset);
    if (retaff != 0) {
        cout_guard.lock();
        cout << "Error setting thread affinity for dedisp thread" << endl;
        cout_guard.unlock();
    }

    cudaCheckError(hipSetDevice(gpuid));
    if (verbose_)
        cout << "Dedisp thread up and running..." << endl;
    int ready{0};
    while(working_) {
        ready = p_mainbuffer->ready();
        if (ready) {
            header_f headerfil;
            headerfil.raw_file = "tastytastytest";
            headerfil.source_name = "J1641-45";
            headerfil.az = 0.0;
            headerfil.dec = 0.0;
            headerfil.fch1 = config_.ftop;
            headerfil.foff = config_.foff;
            headerfil.ra = 0.0;
            headerfil.rdm = 0.0;
            headerfil.tsamp = config_.tsamp;
            // TODO: this totally doesn't work when something is skipped
            headerfil.tstart = get_mjd(start_time.start_epoch, start_time.start_second + gulps_sent * config_.gulp * config_.tsamp);
            headerfil.za = 0.0;
            headerfil.data_type = 1;
            headerfil.ibeam = beamno;
            headerfil.machine_id = 2;
            headerfil.nbeams = 1;
            headerfil.nbits = 32;
            headerfil.nchans = config_.filchans;
            headerfil.nifs = 1;
            headerfil.telescope_id = 2;

            if (verbose_)
                cout << ready - 1 << " buffer ready " << endl;
            p_mainbuffer->send(d_dedisp, ready, mystreams[dstream], (gulps_sent % 2));
            //working = false;
            p_mainbuffer->dump((gulps_sent % 2), headerfil, config_.outdir);
            gulps_sent++;
            //working = false;
        } else {
            std::this_thread::yield();
        }
    }
}

void GPUpool::receive_thread(int ii)
{
    cpu_set_t cpuset;
    CPU_ZERO(&cpuset);
    CPU_SET((int)(poolid_) * 8 + 1 + nostreams + (int)(ii / 3), &cpuset);
    int retaff = pthread_setaffinity_np(receive_threads[ii].native_handle(), sizeof(cpu_set_t), &cpuset);
    if (retaff != 0) {
        cout_guard.lock();
        cerr << "Error setting thread affinity for receive thread on port " << 17000 + ii << endl;
        cout_guard.unlock();
    }

    if (verbose_) {
        cout_guard.lock();
        cout << "Receive thread on port " << 17000 + ii << " running on CPU " << sched_getcpu() << endl;
        cout_guard.unlock();
    }

    sockaddr_storage their_addr;
    memset(&their_addr, 0, sizeof(their_addr));
    socklen_t addr_len;
    memset(&addr_len, 0, sizeof(addr_len));

    const int pack_per_worker_buf = pack_per_buf / nostreams;
    int numbytes{0};
    short fpga{0};
    short bufidx{0};
    // this will always be an integer
    int frame{0};
    int ref_s{0};
    int packcount{0};
    int group{0};

    if (ii == 0) {
        unsigned char *temp_buf = rec_bufs[0];
        numbytes = recvfrom(sfds[ii], rec_bufs[ii], BUFLEN - 1, 0, (struct sockaddr*)&their_addr, &addr_len);
        start_time.start_epoch = (int)(temp_buf[12] >> 2);
        start_time.start_second = (int)(temp_buf[3] | (temp_buf[2] << 8) | (temp_buf[1] << 16) | ((temp_buf[0] & 0x3f) << 24));
        beamno = (int)(temp_buf[23] | (temp_buf[22] << 8));
    }
    // TODO: wait until frame = 0, i.e. we start recording at the 27s boundary
    std::this_thread::sleep_for(std::chrono::seconds(1));

    while (true) {
        if ((numbytes = recvfrom(sfds[ii], rec_bufs[ii], BUFLEN - 1, 0, (struct sockaddr*)&their_addr, &addr_len)) == -1) {
            cout_guard.lock();
            cerr << "Error of recvfrom on port " << 17100 + ii << endl;
            cerr << "Errno " << errno << endl;
            cout_guard.unlock();
        }
        if (numbytes == 0)
            continue;
        frame = (int)(rec_bufs[ii][7] | (rec_bufs[ii][6] << 8) | (rec_bufs[ii][5] << 16) | (rec_bufs[ii][4] << 24));
        if (frame == 0) {
            //cout << 17100 + ii << " starts recording" << endl;
            break;
        }
    }

    while(working_) {
        if ((numbytes = recvfrom(sfds[ii], rec_bufs[ii], BUFLEN - 1, 0, (struct sockaddr*)&their_addr, &addr_len)) == -1) {
            cout_guard.lock();
            cerr << "Error of recvfrom on port " << 17100 + ii << endl;
            cerr << "Errno " << errno << endl;
            cout_guard.unlock();
        }
        if (numbytes == 0)
            continue;
        ref_s = (int)(rec_bufs[ii][3] | (rec_bufs[ii][2] << 8) | (rec_bufs[ii][1] << 16) | ((rec_bufs[ii][0] & 0x3f) << 24));
        frame = (int)(rec_bufs[ii][7] | (rec_bufs[ii][6] << 8) | (rec_bufs[ii][5] << 16) | (rec_bufs[ii][4] << 24));
        fpga = ((short)((((struct sockaddr_in*)&their_addr)->sin_addr.s_addr >> 16) & 0xff) - 1) * 6 + ((int)((((struct sockaddr_in*)&their_addr)->sin_addr.s_addr >> 24)& 0xff) - 1) / 2;
        frame = frame + (ref_s - start_time.start_second - 27) / 27 * 250000;

        // looking at how much stuff we are not missing - remove a lot of checking for now
        // TODO: add some mininal checks later anyway
        //if (frame >= 131008) {
        // which half of the buffer to put the data in
        bufidx = ((int)(frame / accumulate) % nostreams) * pack_per_worker_buf;
        // frame position in the half
        bufidx += (frame % accumulate) * 48;
        frame_times[frame % (accumulate * nostreams)] = frame;
        // frequency chunk in the frame
        bufidx += fpga;
        std::copy(rec_bufs[ii] + HEADER, rec_bufs[ii] + BUFLEN, h_pol + (BUFLEN - HEADER) * bufidx);
        //cout << bufidx << endl;
        //cout.flush();
        bufidx_array[bufidx] = true;
        //}
    }
}
