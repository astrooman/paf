#include <cstdlib>
#include <iostream>
#include <queue>
#include <mutex>
#include <random>
#include <thread>
#include <vector>

#include <buffer.hpp>
#include <config.hpp>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <dedisp.h>
#include <DedispPlan.hpp>
#include <pool.hpp>
#include <vdif.hpp>

// Heimdall headers - including might be a bit messy
#include <params.hpp>
#include <pipeline.hpp>

#include <errno.h>
#include <netdb.h>
#include <arpa/inet.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <unistd.h>

using std::cout;
using std::endl;
using std::mutex;
using std::queue;
using std::string;
using std::thread;
using std::vector;

#define PORT "45003"
#define DATA 7168          // 128 time samples, 7 channels per time sample, 64-bit words
#define BUFLEN 7168 + 64   // 8908 bytes for sample block and 64  bytes for header

void *get_addr(sockaddr *sadr)
{
    if (sadr->sa_family == AF_INET) {
        return &(((sockaddr_in*)sadr)->sin_addr);
    }

    return &(((sockaddr_in6*)sadr)->sin6_addr);
}


int main(int argc, char *argv[])
{
    std::string config_file;

    bool test{false};           // don't use test buffer by default
    bool verbose{false};        // don't use verbose mode by default
    unsigned int chunks{32};    // 32 chunks by default - this is just for testing purposes
    unsigned int streamno{4};   // 4 streams by default
    unsigned int beamno{3};     // 3 beams by default
    unsigned int times{4};      // 4 time samples by default
    unsigned int freq{16};       // no frequency averaging by default, at least 8, possibly 16
    // might be 336 / 168 or 384 / 192
    unsigned int nchans{336};   // number of 1MHz channels - might change

    // dedispersion parameters
    double band = 1.185;         // sampling rate for each band in MHz
    double dstart{0.0};
    double dend{4000.0};
    double foff{0.0};
    double ftop{0.0};
    double tsamp = ((double)1.0 / (band * 1e+06) * (double)32.0);
    unsigned int filchans{nchans * 27 / freq};
    unsigned int gulp{131072};  // 2^17, equivalent to ~14s for 108us sampling time

    int *killmask = new int[filchans];

    config_s config;
    default_config(config);

    // too many parameters to load as arguments - use config file
    if (argc >= 2) {
        for (int ii = 0; ii < argc; ii++) {
            if (std::string(argv[ii]) == "--config") {      // configuration file
                ii++;
                config_file = std::string(argv[ii]);
                read_config(config_file, config);
                break;      // configuration file should have everything included
            }
            if (std::string(argv[ii]) == "-c") {      // the number of chunks to process
                ii++;
                config.chunks = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-s") {     // the number of streams to use
                ii++;
                config.streamno = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-b") {     // the number of beams to accept the data from
                ii++;
                config.beamno = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-t") {     // the number of time sample to average
                ii++;
                config.times = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-f") {     // the number of frequency channels to average
                ii++;
                config.freq = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-b") {     // use the test buffer
                config.test = true;
            } else if (std::string(argv[ii]) == "-v") {
                config.verbose = true;
            } else if (std::string(argv[ii]) == "-h") {
                cout << "Options:\n"
                        << "\t -v - use verbose mode\n"
                        << "\t -c - the number of chunks to process\n"
                        << "\t -b - the number of beams to process\n"
                        << "\t -t - the number of time samples to average\n"
                        << "\t -f - the number of frequency channels to average\n"
                        << "\t -s - the number of CUDA streams to use\n"
                        << "\t -b - use the test buffer\n"
                        << "\t -h - print out this message\n"
                        << "\t --config - configuration file\n\n";
                exit(EXIT_SUCCESS);
            }
        }

    }
    // should not take more than 5 seconds
    cout << "Starting up. This may take few seconds..." << endl;
    // tsamp in seconds, ftop and foff in MHz
    DedispPlan dedisp(filchans, tsamp, ftop, foff);
    // width is the expected pulse width in microseconds
    // tol is the smearing tolerance factor between two DM trials
    dedisp.generate_dm_list(dstart, dend, (float)64.0, (float)1.10);
    size_t totsamples = (size_t)gulp + dedisp.get_max_delay();
    unsigned int buffno = (totsamples - 1) / gulp + 1;
    size_t buffsize = buffno * gulp + dedisp.get_max_delay();
    cout << "Will try " << dedisp.get_dm_count() << " DM trials" << endl;
    if (verbose) {
        cout << "Will try " << dedisp.get_dm_count() << " DM trials:\n";
        for (int ii = 0; ii < dedisp.get_dm_count(); ii++)
            cout << *(dedisp.get_dm_list() + ii) << endl;
    }
    if (false)       // switch off for now
        dedisp.set_killmask(killmask);


    // using thread pool will remove the need of checking which stream is used
    // each thread will be associated with a separate stream
    // it will start proceesing the new chunk as soon as possible
    unsigned int batchs{config.beamno * config.nchans};      // # beams * 192 channels
                                            // need to decide how this data will be stored
    unsigned int ffts{32};
    Pool mypool(batchs, ffts, config.times, config.streamno, config.freq, config);

    // networking stuff
    int sfd, numbytes, rv;
    socklen_t addrlen;              // socklen_t has length of at least 32 bits
    addrinfo hints, *servinfo, *p;
    sockaddr_storage their_addr;    // sockaddr_storage is large enough accommodate all supported
                                    //protocol-specific address structures
    char s[INET6_ADDRSTRLEN];       // length of the string form for IPv6
    hipfftComplex *chunkbuf = new hipfftComplex[batchs * ffts * times];
    //unsigned int mempacket = 6144;   // how many bytes per packet to read
    size_t memsize = batchs * ffts * times * sizeof(hipfftComplex);
    //const unsigned int packets = memsize / mempacket;   // const to avoid accidental changes
                                                        // number of packets require to receive
                                                        // one data 'chunk', i.e. the amount of
                                                        // data required to performed filterbanking
                                                        // with averaging for all necessary beams and channels
    // unsigned int packetel = mempacket / sizeof(hipfftComplex);
    unsigned char *inbuf = new unsigned char[BUFLEN];
    memset(&hints, 0, sizeof(hints));
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_DGRAM;
    hints.ai_flags = AI_PASSIVE;    // allows to use NULL in getaddrinfo

    cout << "Will process " << memsize / 1024.0 << "KB chunks\n"
		<< "Divided into " << packets << " 6144B packets\n";

    if((rv = getaddrinfo(NULL, PORT, &hints, &servinfo)) != 0) {
        cout << "getaddrinfo error " << gai_strerror(rv) << endl;
        exit(EXIT_FAILURE);
    }

    // loop through the linked list and try binding to the first possible socket
    for (p = servinfo; p != NULL; p = p->ai_next) {
        if((sfd = socket(p->ai_family, p->ai_socktype, p->ai_protocol)) == -1) {
            perror("socket ");
            continue;
        }

        if(bind(sfd, p->ai_addr, p->ai_addrlen) == -1) {
            close(sfd);
            perror("bind ");
            continue;
        }
        break;
    }
    // didn't bind to anything
    if (p == NULL) {
        cout << "error: failed to bind the socket\n";
        exit(EXIT_FAILURE);
    }

    freeaddrinfo(servinfo);     // no longer need it
    cout << "Waiting to receive from the server...\n";

    int chunkno{0};

    header_s head;

    int polsize = nchans * times * 32;

    hipfftComplex *pola = new hipfftComplex[polsize];
    hipfftComplex *polb = new hipfftComplex[polsize];

    int previous_frame  = -1;

    // proper data receiving
    while(true) {

        numbytes = recvfrom(sfd, inbuf, BUFLEN, 0, (struct sockaddr*)&their_addr, &addrlen);

        // assume last packet will have 0 bytes
        if(!numbytes)
            break;
        get_header(inbuf, head);
        my_pool.get_data(inbuf, head.frame_no);
    }

    // while(chunkno < chunks) {
    //     // will only receive 6 or 7 channels in one packet
    //     // will have to stitch them together
    //     for (unsigned int packetno  = 0; packetno < packets; packetno++) {
    //         if((numbytes = recvfrom(sfd, inbuf, mempacket, 0, (struct sockaddr*)&their_addr, &addrlen)) == -1 ) {
    //             cout << "error recvfrom" << endl;
    //             exit(EXIT_FAILURE);
    //         }
    //         // get the vdif header and strip it off the data
    //         get_header(inbuf, head);
    //         //cout << "Received packet " << packetno << " with " << numbytes << " bytes\n";
    //         //cout.flush();
    //         // I am not happy with the amount of copying done here and below
    //         // COMMENTED OUT FOR COMPILATION - READING VDIF FILES WILL BE SORTED OUT
    //         //std::copy(inbuf, inbuf + packetel, chunkbuf + packetno * packetel);
    //     }
    //
    //     mypool.add_data(chunkbuf);
    //     //cout << "Received chunk " << chunkno << endl;
    //     //cout.flush();
    //     chunkno++;
    //     // will send 0 bytes as a last packet to end the loop
    //     if(!numbytes)
    //         break;
    //     inet_ntop(their_addr.ss_family, get_addr((sockaddr*)&their_addr), s, sizeof(s));
    // }
    //
    // if(test) {
    //
    //     cout << "Test buffer\n";
    //     cout.flush();
    //     // sleep just in case processing is slow
    //     std::this_thread::sleep_for(std::chrono::seconds(1));
    //
    //     hipfftComplex *testbuf = new hipfftComplex[batchs * ffts * times * chunks];
    //
    //     unsigned int seed = std::chrono::system_clock::now().time_since_epoch().count();
    //     std::mt19937_64 bufeng{seed};
    //     std::normal_distribution<float> bufdis(0.0, 1.0);
    //
    //     cout << "Filling the test array...\n";
    //
    //     for (int ii = 0; ii < batchs * ffts * times * chunks; ii++) {
    //         testbuf[ii].x = bufdis(bufeng);
    //         testbuf[ii].y = bufdis(bufeng);
    //     }
    //
    //     chunkno = 0;
    //     while(chunkno < chunks) {
    //         mypool.add_data(testbuf + chunkno * batchs * ffts * times);
    //         chunkno++;
    //     }
    //
    // }

    std::this_thread::sleep_for(std::chrono::seconds(2));

    hipDeviceReset();

    return 0;
}
