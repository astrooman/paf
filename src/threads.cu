#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <queue>
#include <mutex>
#include <random>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <dedisp.h>
#include <DedispPlan.hpp>
#include <vdif_head.hpp>

#include <errno.h>
#include <netdb.h>
#include <arpa/inet.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <unistd.h>

using std::cout;
using std::endl;
using std::mutex;
using std::queue;
using std::string;
using std::thread;
using std::vector;

#define PORT "45003"
#define SINGLE_GULP 131027      // number of time samples per single-pulse detection data chunk

__global__ void poweradd(hipfftComplex *in, float *out, unsigned int jump);

void *get_addr(sockaddr *sadr)
{
    if (sadr->sa_family == AF_INET) {
        return &(((sockaddr_in*)sadr)->sin_addr);
    }

    return &(((sockaddr_in6*)sadr)->sin6_addr);
}
// make a template to support dfferent dedisp input types
template <class T>
class Buffer
{
    private:
        size_t start;
        size_t end;
        const size_t size;
        T *d_buf;
    protected:

    public:
        Buffer(size_t size);
        ~Buffer(void);

        void write(T *data);
        // add deleted copy, move, etc constructors
};

template<class T>
Buffer<T>::Buffer(size_t size) : size(size)
{
    start = 0;
    end = 0;
    hipMalloc((void**)&d_buf, size * sizeof(T));
}

template<class T>
Buffer<T>::~Buffer()
{
    end = 0;
    hipFree(d_buf);
}

template<class T>
void Buffer<T>::write(T *d_data, unsigned int amount)
{
    end = end + amount;
    hipMemcpy
}

class Pool
{
    private:
        bool working;
        // const to be safe
        const unsigned int batchsize;
        const unsigned int fftsize;
        const unsigned int timesamp;
        const unsigned int streamno;
        const unsigned int freqavg;
        // one buffer
        unsigned int bufs;
        unsigned int bufsize;
        unsigned int bufmem;
        // buffer for all streams together
        unsigned int totsize;
        unsigned int totmem;
        // GPU and thread stuff
        hipfftComplex *h_in, *d_in;
        float *h_out, *d_out;
        int sizes[1];
        int avt;
        hipStream_t *mystreams;
        hipfftHandle *myplans;
        mutex datamutex;
        mutex workmutex;
        unsigned int nthreads;
        unsigned int nblocks;
        // containers
        // use queue as FIFO needed
        queue<vector<hipfftComplex>> mydata;
        vector<thread> mythreads;
    protected:

    public:
        Pool(unsigned int bs, unsigned int fs, unsigned int ts, unsigned int sn, unsigned int fr, unsigned int bn);
        ~Pool(void);
        // add deleted copy, move, etc constructors
        void add_data(hipfftComplex *buffer);
        void minion(int stream);
};

Pool::Pool(unsigned int bs, unsigned int fs, unsigned int ts, unsigned int sn, unsigned int fr, unsigned int bn) : batchsize(bs),
                                                                fftsize(fs),
                                                                timesamp(ts),
                                                                working(true),
                                                                streamno(sn),
                                                                freqavg(fr),
                                                                bufs(bn),
                                                                nthreads(256)
{

    avt = min(streamno,thread::hardware_concurrency());
    bufsize = fftsize * batchsize * timesamp;
    bufmem = bufsize * sizeof(hipfftComplex);
    totsize = bufsize * avt;
    totmem = bufmem * avt;
    // / 2 as interested in time averaged output
    nblocks = (bufsize / 2 - 1 ) / nthreads + 1;

    sizes[0] = (int)fftsize;
    // want as many streams and plans as there will be threads
    // every thread will be associated with its own stream
    mystreams = new hipStream_t[avt];
    myplans = new hipfftHandle[avt];

    hipHostAlloc((void**)&h_in, totsize * sizeof(hipfftComplex), hipHostMallocDefault);
    hipHostAlloc((void**)&h_out, totsize / 2 * sizeof(float), hipHostMallocDefault);
    hipMalloc((void**)&d_in, totsize * sizeof(hipfftComplex));
    hipMalloc((void**)&d_out, totsize * avt / 2 * sizeof(float));

    for (int ii = 0; ii < avt; ii++) {
        hipStreamCreate(&mystreams[ii]);
        hipfftPlanMany(&myplans[ii], 1, sizes, NULL, 1, fftsize, NULL, 1, fftsize, HIPFFT_C2C, batchsize);
        hipfftSetStream(myplans[ii], mystreams[ii]);
        // need to meet requirements for INVOKE(f, t1, t2, ... tn)
        // (t1.*f)(t2, ... tn) when f is a pointer to a member function of class T
        // and t1 is an object of type T or a reference to an object of type T
        // or a reference to an object of a type derived from T (C++14 §20.9.2)
        mythreads.push_back(thread(&Pool::minion, this, ii));

    }
}

Pool::~Pool(void)
{
    working = false;
    // join the threads so main() thread will wait until all 4 last pieces are processed
    for (int ii = 0; ii < avt; ii++)
        mythreads[ii].join();
}

void Pool::add_data(hipfftComplex *buffer)
{
    std::lock_guard<mutex> addguard(datamutex);
    // that has to have a mutex
    mydata.push(vector<hipfftComplex>(buffer, buffer + bufsize));
    //cout << "Data added\n";
    //cout.flush();
}

void Pool::minion(int stream)
{
    cout << "Starting thread associated with stream " << stream << endl << endl;
    cout.flush();

    unsigned int skip = stream * bufsize;
    unsigned int outmem = bufsize / 2 * sizeof(float);

    while(working) {
        // need to protect if with mutex
        // current mutex implementation is a big ugly, but just need a dirty hack
        // will write a new, thread-safe queue implementation
        datamutex.lock();
        if(!mydata.empty()) {
            std::copy((mydata.front()).begin(), (mydata.front()).end(), h_in + skip);
            mydata.pop();
            datamutex.unlock();
	        //cout << "Stream " << stream << " got the data\n";
	        //cout.flush();
            if(hipMemcpyAsync(d_in + skip, h_in + skip, bufmem, hipMemcpyHostToDevice, mystreams[stream]) != hipSuccess) {
		        cout << "HtD copy error on stream " << stream << " " << hipGetErrorString(hipGetLastError()) << endl;
		        cout.flush();
	        }
            if(hipfftExecC2C(myplans[stream], d_in + skip, d_in + skip, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
		          cout << "Error in FFT execution\n";
            poweradd<<<nblocks, nthreads, 0, mystreams[stream]>>>(d_in + skip, d_out + skip / 2, fftsize * batchsize);
            if(hipMemcpyAsync(h_out + skip / 2, d_out + skip / 2, outmem, hipMemcpyDeviceToHost, mystreams[stream]) != hipSuccess) {
		        cout << "DtH copy error on stream " << stream << " " << hipGetErrorString(hipGetLastError()) << endl;
		        cout.flush();
	        }
            hipDeviceSynchronize();
        } else {
	        datamutex.unlock();
            std::this_thread::yield();
        }
    }
}

__global__ void poweradd(hipfftComplex *in, float *out, unsigned int jump)
{
    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
	// offset introduced - can cause some slowing down
	int idx2 = blockIdx.x * blockDim.x + threadIdx.x + jump;

    if (idx1 < jump) {      // half of the input data
        float power1 = in[idx1].x * in[idx1].x + in[idx1].y * in[idx1].y;
        float power2 = in[idx2].x * in[idx2].x + in[idx2].y * in[idx2].y;
        out[idx1] = (power1 + power2) / 2.0;
    }
}

int main(int argc, char *argv[])
{
    std::string config_file;

    bool test{false};           // don't use test buffer by default
    bool verbose{false};        // don't use verbose mode by default
    unsigned int chunks{32};    // 32 chunks by default - this is just for testing purposes
    unsigned int streamno{4};   // 4 streams by default
    unsigned int beamno{3};     // 3 beams by default
    unsigned int times{4};      // 2 time samples by default
    unsigned int freq{8};       // no frequency averaging by default, at least 8, possibly 16
    // might be 336 / 168 or 384 / 192
    unsigned int nchans{192};   // number of 1MHz channels - might change

    // dedispersion parameters
    double band = 1.185;         // sampling rate for each band in MHz
    double dstart{0.0};
    double dend{4000.0};
    double foff{0.0};
    double ftop{0.0};
    double tsamp = ((double)1.0 / (band * 1e+06) * (double)32.0);
    unsigned int filchans{nchans * 27 / freq};
    unsigned int gulp{131072};  // 2^17, equivalent to ~14s for 108us sampling time

    int *killmask = new int[filchans];

    // too many parameters to load as arguments - use config file
    if (argc >= 2) {
        for (int ii = 0; ii < argc; ii++) {
            if (std::string(argv[ii]) == "--config") {      // configuration file
                ii++;
                config_file = std::string(argv[ii]);
                break;      // configuration file should have everything included
            }
            if (std::string(argv[ii]) == "-c") {      // the number of chunks to process
                ii++;
                chunks = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-s") {     // the number of streams to use
                ii++;
                streamno = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-b") {     // the number of beams to accept the data from
                ii++;
                beamno = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-t") {     // the number of time sample to average
                ii++;
                times = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-f") {     // the number of frequency channels to average
                ii++;
                freq = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-b") {     // use the test buffer
                test = true;
            } else if (std::string(argv[ii]) == "-v") {
                verbose = true;
            } else if (std::string(argv[ii]) == "-h") {
                cout << "Options:\n"
                        << "\t -v - use verbose mode\n"
                        << "\t -c - the number of chunks to process\n"
                        << "\t -b - the number of beams to process\n"
                        << "\t -t - the number of time samples to average\n"
                        << "\t -f - the number of frequency channels to average\n"
                        << "\t -s - the number of CUDA streams to use\n"
                        << "\t -b - use the test buffer\n"
                        << "\t -h - print out this message\n"
                        << "\t --config - configuration file\n\n";
                exit(EXIT_SUCCESS);
            }
        }

    }
    // should not take more than 5 seconds
    cout << "Starting up. This may take few seconds..." << endl;
    // tsamp in seconds, ftop and foff in MHz
    DedispPlan dedisp(filchans, tsamp, ftop, foff);
    // width is the expected pulse width in microseconds
    // tol is the smearing tolerance factor between two DM trials
    dedisp.generate_dm_list(dstart, dend, (float)64.0, (float)1.10);
    size_t buffsize = (size_t)gulp + dedisp.get_max_delay();
    unsigned int buffno = (buffsize - 1) / gulp + 1;
    cout << "Will try " << dedisp.get_dm_count() << " DM trials" << endl;
    if (verbose) {
        cout << "Will try " << dedisp.get_dm_count() << " DM trials:\n";
        for (int ii = 0; ii < dedisp.get_dm_count(); ii++)
            cout << *(dedisp.get_dm_list() + ii) << endl;
    }
    if (false)       // switch off for now
        dedisp.set_killmask(killmask);
    // everything should be ready for dedispersion after this point

    // using thread pool will remove the need of checking which stream is used
    // each thread will be associated with a separate stream
    // it will start proceesing the new chunk as soon as possible
    unsigned int batchs{beamno * nchans};      // # beams * 192 channels
                                            // need to decide how this data will be stored
    unsigned int ffts{32};
    Pool mypool(batchs, ffts, times, streamno, freq, buffno);

    // networking stuff
    int sfd, numbytes, rv;
    socklen_t addrlen;              // socklen_t has length of at least 32 bits
    addrinfo hints, *servinfo, *p;
    sockaddr_storage their_addr;    // sockaddr_storage is large enough accommodate all supported
                                    //protocol-specific address structures
    char s[INET6_ADDRSTRLEN];       // length of the string form for IPv6
    hipfftComplex *chunkbuf = new hipfftComplex[batchs * ffts * times];
    unsigned int mempacket = 6144;   // how many bytes per packet to read
    size_t memsize = batchs * ffts * times * sizeof(hipfftComplex);
    const unsigned int packets = memsize / mempacket;   // const to avoid accidental changes
                                                        // number of packets require to receive
                                                        // one data 'chunk', i.e. the amount of
                                                        // data required to performed filterbanking
                                                        // with averaging for all necessary beams and channels
    unsigned int packetel = mempacket / sizeof(hipfftComplex);
    hipfftComplex *inbuf = new hipfftComplex[packetel];
    memset(&hints, 0, sizeof(hints));
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_DGRAM;
    hints.ai_flags = AI_PASSIVE;    // allows to use NULL in getaddrinfo

    cout << "Will process " << memsize / 1024.0 << "KB chunks\n"
		<< "Divided into " << packets << " 6144B packets\n";

    if((rv = getaddrinfo(NULL, PORT, &hints, &servinfo)) != 0) {
        cout << "getaddrinfo error " << gai_strerror(rv) << endl;
        exit(EXIT_FAILURE);
    }

    // loop through the linked list and try binding to the first possible socket
    for (p = servinfo; p != NULL; p = p->ai_next) {
        if((sfd = socket(p->ai_family, p->ai_socktype, p->ai_protocol)) == -1) {
            perror("socket ");
            continue;
        }

        if(bind(sfd, p->ai_addr, p->ai_addrlen) == -1) {
            close(sfd);
            perror("bind ");
            continue;
        }
        break;
    }
    // didn't bind to anything
    if (p == NULL) {
        cout << "error: failed to bind the socket\n";
        exit(EXIT_FAILURE);
    }

    freeaddrinfo(servinfo);     // no longer need it
    cout << "Waiting to receive from the server...\n";

    int chunkno{0};

    header_s head;

    while(chunkno < chunks) {

        for (unsigned int packetno  = 0; packetno < packets; packetno++) {
            if((numbytes = recvfrom(sfd, inbuf, mempacket, 0, (struct sockaddr*)&their_addr, &addrlen)) == -1 ) {
                cout << "error recvfrom" << endl;
                exit(EXIT_FAILURE);
            }
            // get the vdif header and strip it off the data
            get_header(inbuf, head);
            //cout << "Received packet " << packetno << " with " << numbytes << " bytes\n";
            //cout.flush();
            // I am not happy with the amount of copying done here and below
            std::copy(inbuf, inbuf + packetel, chunkbuf + packetno * packetel);
        }

        mypool.add_data(chunkbuf);
        //cout << "Received chunk " << chunkno << endl;
        //cout.flush();
        chunkno++;
        // will send 0 bytes as a last packet to end the loop
        if(!numbytes)
            break;
        inet_ntop(their_addr.ss_family, get_addr((sockaddr*)&their_addr), s, sizeof(s));
    }

    if(test) {

        cout << "Test buffer\n";
        cout.flush();
        // sleep just in case processing is slow
        std::this_thread::sleep_for(std::chrono::seconds(1));

        hipfftComplex *testbuf = new hipfftComplex[batchs * ffts * times * chunks];

        unsigned int seed = std::chrono::system_clock::now().time_since_epoch().count();
        std::mt19937_64 bufeng{seed};
        std::normal_distribution<float> bufdis(0.0, 1.0);

        cout << "Filling the test array...\n";

        for (int ii = 0; ii < batchs * ffts * times * chunks; ii++) {
            testbuf[ii].x = bufdis(bufeng);
            testbuf[ii].y = bufdis(bufeng);
        }

        chunkno = 0;
        while(chunkno < chunks) {
            mypool.add_data(testbuf + chunkno * batchs * ffts * times);
            chunkno++;
        }

    }

    std::this_thread::sleep_for(std::chrono::seconds(2));

    hipDeviceReset();

    return 0;
}
