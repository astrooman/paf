#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <queue>
#include <mutex>
#include <random>
#include <thread>
#include <vector>

#include <buffer.hpp>
#include <config.hpp>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <dedisp.h>
#include <DedispPlan.hpp>
#include <vdif.hpp>

// Heimdall headers - including might be a bit messy
#include <params.hpp>

#include <errno.h>
#include <netdb.h>
#include <arpa/inet.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <unistd.h>

using std::cout;
using std::endl;
using std::mutex;
using std::queue;
using std::string;
using std::thread;
using std::vector;

#define PORT "45003"
#define SINGLE_GULP 131027      // number of time samples per single-pulse detection data chunk

__global__ void poweradd(hipfftComplex *in, unsigned char *out, unsigned int jump);

void *get_addr(sockaddr *sadr)
{
    if (sadr->sa_family == AF_INET) {
        return &(((sockaddr_in*)sadr)->sin_addr);
    }

    return &(((sockaddr_in6*)sadr)->sin6_addr);
}

class Pool
{
    private:
        // that can be anything, depending on how many output bits we decide to use
        Buffer<unsigned char> mainbuffer;
        DedispPlan dedisp;
        hd_params params;

        bool working;
        // const to be safe
        const unsigned int batchsize;
        const unsigned int fftsize;
        const unsigned int timesamp;
        const unsigned int streamno;
        const unsigned int freqavg;
        // one buffer
        unsigned int filsize;
        unsigned int bufmem;
        // buffer for all streams together
        unsigned int totsize;
        unsigned int totmem;
        // GPU and thread stuff
        unsigned char *d_dedisp;
        unsigned char *d_search;
        hipfftComplex *h_in, *d_in;
        unsigned char *h_out, *d_out;
        int sizes[1];
        int avt;
        hipStream_t *mystreams;
        hipfftHandle *myplans;
        mutex datamutex;
        mutex workmutex;
        unsigned int nthreads;
        unsigned int nblocks;
        // containers
        // use queue as FIFO needed
        queue<vector<hipfftComplex>> mydata;
        vector<thread> mythreads;
        unsigned int buffno;
        size_t totsamples;
    protected:

    public:
        Pool(unsigned int bs, unsigned int fs, unsigned int ts, unsigned int sn, unsigned int fr, config_s config);
        ~Pool(void);
        // add deleted copy, move, etc constructors
        void add_data(hipfftComplex *buffer);
        void dedisp_thread(int dstream);
        void minion(int stream);
        void search_thread(int sstream);
};

Pool::Pool(unsigned int bs, unsigned int fs, unsigned int ts, unsigned int sn, unsigned int fr, config_s config) : batchsize(bs),
                                                                fftsize(fs),
                                                                timesamp(ts),
                                                                working(true),
                                                                streamno(sn),
                                                                freqavg(fr),
                                                                nthreads(256),
                                                                mainbuffer(),
                                                                dedisp(config.filchans, config.tsamp, config.ftop, config.foff)
{
    // streamno for filterbank and additional 2 for dedispersion and single pulse search
    avt = min(streamno + 2,thread::hardware_concurrency());

    if(config.verbose)
        cout << "Will create " << avt << " CUDA streams\n";

    // width is the expected pulse width in microseconds
    // tol is the smearing tolerance factor between two DM trials
    dedisp.generate_dm_list(config.dstart, config.dend, (float)64.0, (float)1.10);
    if (config.verbose) {
        cout << "Will try " << dedisp.get_dm_count() << " DM trials:\n";
        for (int ii = 0; ii < dedisp.get_dm_count(); ii++)
            cout << *(dedisp.get_dm_list() + ii) << endl;
    }

    totsamples = (size_t)config.gulp + dedisp.get_max_delay();
    buffno = (totsamples - 1) / config.gulp + 1;
    size_t buffsize = buffno * config.gulp + dedisp.get_max_delay();
    mainbuffer.allocate(buffno, dedisp.get_max_delay(), config.gulp, buffsize);
    //if (false)       // switch off for now
    //    dedisp.set_killmask(killmask);
    // everything should be ready for dedispersion after this point

    set_search_params(&params, config);
    // everything should be ready for single pulse search after this point

    filsize = fftsize * batchsize * timesamp;
    bufmem = filsize * sizeof(hipfftComplex);
    totsize = filsize * avt;
    totmem = bufmem * avt;
    // / 2 as interested in time averaged output
    nblocks = (filsize / 2 - 1 ) / nthreads + 1;

    sizes[0] = (int)fftsize;
    // want as many streams and plans as there will be threads
    // every thread will be associated with its own stream
    mystreams = new hipStream_t[avt];
    myplans = new hipfftHandle[avt];

    hipHostAlloc((void**)&h_in, totsize * sizeof(hipfftComplex), hipHostMallocDefault);
    hipHostAlloc((void**)&h_out, totsize / 2 * sizeof(float), hipHostMallocDefault);
    hipMalloc((void**)&d_in, totsize * sizeof(hipfftComplex));
    hipMalloc((void**)&d_out, totsize * avt / 2 * sizeof(unsigned char));
    // change this later to deal with any input type;
    hipMalloc((void**)&d_dedisp, totsamples * sizeof(unsigned char));
    hipMalloc((void**)&d_search, config.gulp * dedisp.get_dm_count() * sizeof(unsigned char));
    // here only launch threads that will take care of filterbank
    for (int ii = 0; ii < avt - 2; ii++) {
        hipStreamCreate(&mystreams[ii]);
        hipfftPlanMany(&myplans[ii], 1, sizes, NULL, 1, fftsize, NULL, 1, fftsize, HIPFFT_C2C, batchsize);
        hipfftSetStream(myplans[ii], mystreams[ii]);
        // need to meet requirements for INVOKE(f, t1, t2, ... tn)
        // (t1.*f)(t2, ... tn) when f is a pointer to a member function of class T
        // and t1 is an object of type T or a reference to an object of type T
        // this is t1 and &Pool::minion is a pointer to a member function of class T
        // or a reference to an object of a type derived from T (C++14 §20.9.2)
        mythreads.push_back(thread(&Pool::minion, this, ii));
    }

    // dedisp thread
    hipStreamCreate(&mystreams[avt-2]);
    mythreads.push_back(thread(&Pool::dedisp_thread, this, avt-2));
    // single pulse thread
    hipStreamCreate(&mystreams[avt-1]);
    mythreads.push_back(thread(&Pool::search_thread, this, avt-1));
}

Pool::~Pool(void)
{
    working = false;
    // join the threads so main() thread will wait until all 4 last pieces are processed
    for (int ii = 0; ii < avt; ii++)
        mythreads[ii].join();
}

void Pool::add_data(hipfftComplex *buffer)
{
    std::lock_guard<mutex> addguard(datamutex);
    // that has to have a mutex
    mydata.push(vector<hipfftComplex>(buffer, buffer + filsize));
    //cout << "Data added\n";
    //cout.flush();
}

void Pool::minion(int stream)
{
    cout << "Starting thread associated with stream " << stream << endl << endl;
    cout.flush();

    unsigned int skip = stream * filsize;
    unsigned int outmem = filsize / 2 * sizeof(float);

    while(working) {
        // need to protect if with mutex
        // current mutex implementation is a big ugly, but just need a dirty hack
        // will write a new, thread-safe queue implementation
        unsigned int index{0};       // index will be used to distinguish between time samples
        datamutex.lock();
        if(!mydata.empty()) {
            std::copy((mydata.front()).begin(), (mydata.front()).end(), h_in + skip);
            mydata.pop();
            datamutex.unlock();
	        //cout << "Stream " << stream << " got the data\n";
	        //cout.flush();
            if(hipMemcpyAsync(d_in + skip, h_in + skip, bufmem, hipMemcpyHostToDevice, mystreams[stream]) != hipSuccess) {
		        cout << "HtD copy error on stream " << stream << " " << hipGetErrorString(hipGetLastError()) << endl;
		        cout.flush();
	        }
            if(hipfftExecC2C(myplans[stream], d_in + skip, d_in + skip, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
		          cout << "Error in FFT execution\n";
            poweradd<<<nblocks, nthreads, 0, mystreams[stream]>>>(d_in + skip, d_out + skip / 2, fftsize * batchsize);
            if(hipMemcpyAsync(h_out + skip / 2, d_out + skip / 2, outmem, hipMemcpyDeviceToHost, mystreams[stream]) != hipSuccess) {
		        cout << "DtH copy error on stream " << stream << " " << hipGetErrorString(hipGetLastError()) << endl;
		        cout.flush();
	        }
            mainbuffer.write(d_out, index, filsize / 2, mystreams[stream]);
            hipDeviceSynchronize();
        } else {
	        datamutex.unlock();
            std::this_thread::yield();
        }
    }
}

void Pool::dedisp_thread(int dstream)
{
    int ready = mainbuffer.ready();
    if (ready) {
        mainbuffer.send(d_dedisp, ready, mystreams[dstream]);
        // TO DO: include data member with the number of gulps already dedispersed
        cout << "Dedispersing gulp " << endl;
        dedisp.execute(totsamples, d_dedisp, 8, d_search, 8, DEDISP_DEVICE_POINTERS);
    } else {
        std::this_thread::yield();
    }
}

void Pool::search_thread(int sstream)
{
    if () {
        // TO DO: same as with the dedispersed gulps
        cout << "Searching in the gulp " << endl;
  } else {
        std::this_thread::yield();
  }
}

__global__ void poweradd(hipfftComplex *in, unsigned char *out, unsigned int jump)
{
    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
	// offset introduced - can cause some slowing down
	int idx2 = blockIdx.x * blockDim.x + threadIdx.x + jump;

    if (idx1 < jump) {      // half of the input data
        float power1 = in[idx1].x * in[idx1].x + in[idx1].y * in[idx1].y;
        float power2 = in[idx2].x * in[idx2].x + in[idx2].y * in[idx2].y;
        out[idx1] = (power1 + power2) / 2.0;
    }
}

int main(int argc, char *argv[])
{
    std::string config_file;

    bool test{false};           // don't use test buffer by default
    bool verbose{false};        // don't use verbose mode by default
    unsigned int chunks{32};    // 32 chunks by default - this is just for testing purposes
    unsigned int streamno{4};   // 4 streams by default
    unsigned int beamno{3};     // 3 beams by default
    unsigned int times{4};      // 2 time samples by default
    unsigned int freq{8};       // no frequency averaging by default, at least 8, possibly 16
    // might be 336 / 168 or 384 / 192
    unsigned int nchans{192};   // number of 1MHz channels - might change

    // dedispersion parameters
    double band = 1.185;         // sampling rate for each band in MHz
    double dstart{0.0};
    double dend{4000.0};
    double foff{0.0};
    double ftop{0.0};
    double tsamp = ((double)1.0 / (band * 1e+06) * (double)32.0);
    unsigned int filchans{nchans * 27 / freq};
    unsigned int gulp{131072};  // 2^17, equivalent to ~14s for 108us sampling time

    int *killmask = new int[filchans];

    config_s config;
    default_config(config);

    // too many parameters to load as arguments - use config file
    if (argc >= 2) {
        for (int ii = 0; ii < argc; ii++) {
            if (std::string(argv[ii]) == "--config") {      // configuration file
                ii++;
                config_file = std::string(argv[ii]);
                read_config(config_file, config);
                break;      // configuration file should have everything included
            }
            if (std::string(argv[ii]) == "-c") {      // the number of chunks to process
                ii++;
                config.chunks = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-s") {     // the number of streams to use
                ii++;
                config.streamno = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-b") {     // the number of beams to accept the data from
                ii++;
                config.beamno = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-t") {     // the number of time sample to average
                ii++;
                config.times = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-f") {     // the number of frequency channels to average
                ii++;
                config.freq = atoi(argv[ii]);
            } else if (std::string(argv[ii]) == "-b") {     // use the test buffer
                config.test = true;
            } else if (std::string(argv[ii]) == "-v") {
                config.verbose = true;
            } else if (std::string(argv[ii]) == "-h") {
                cout << "Options:\n"
                        << "\t -v - use verbose mode\n"
                        << "\t -c - the number of chunks to process\n"
                        << "\t -b - the number of beams to process\n"
                        << "\t -t - the number of time samples to average\n"
                        << "\t -f - the number of frequency channels to average\n"
                        << "\t -s - the number of CUDA streams to use\n"
                        << "\t -b - use the test buffer\n"
                        << "\t -h - print out this message\n"
                        << "\t --config - configuration file\n\n";
                exit(EXIT_SUCCESS);
            }
        }

    }
    // should not take more than 5 seconds
    cout << "Starting up. This may take few seconds..." << endl;
    // tsamp in seconds, ftop and foff in MHz
    DedispPlan dedisp(filchans, tsamp, ftop, foff);
    // width is the expected pulse width in microseconds
    // tol is the smearing tolerance factor between two DM trials
    dedisp.generate_dm_list(dstart, dend, (float)64.0, (float)1.10);
    size_t totsamples = (size_t)gulp + dedisp.get_max_delay();
    unsigned int buffno = (totsamples - 1) / gulp + 1;
    size_t buffsize = buffno * gulp + dedisp.get_max_delay();
    cout << "Will try " << dedisp.get_dm_count() << " DM trials" << endl;
    if (verbose) {
        cout << "Will try " << dedisp.get_dm_count() << " DM trials:\n";
        for (int ii = 0; ii < dedisp.get_dm_count(); ii++)
            cout << *(dedisp.get_dm_list() + ii) << endl;
    }
    if (false)       // switch off for now
        dedisp.set_killmask(killmask);


    // using thread pool will remove the need of checking which stream is used
    // each thread will be associated with a separate stream
    // it will start proceesing the new chunk as soon as possible
    unsigned int batchs{config.beamno * config.nchans};      // # beams * 192 channels
                                            // need to decide how this data will be stored
    unsigned int ffts{32};
    Pool mypool(batchs, ffts, config.times, config.streamno, config.freq, config);

    // networking stuff
    int sfd, numbytes, rv;
    socklen_t addrlen;              // socklen_t has length of at least 32 bits
    addrinfo hints, *servinfo, *p;
    sockaddr_storage their_addr;    // sockaddr_storage is large enough accommodate all supported
                                    //protocol-specific address structures
    char s[INET6_ADDRSTRLEN];       // length of the string form for IPv6
    hipfftComplex *chunkbuf = new hipfftComplex[batchs * ffts * times];
    unsigned int mempacket = 6144;   // how many bytes per packet to read
    size_t memsize = batchs * ffts * times * sizeof(hipfftComplex);
    const unsigned int packets = memsize / mempacket;   // const to avoid accidental changes
                                                        // number of packets require to receive
                                                        // one data 'chunk', i.e. the amount of
                                                        // data required to performed filterbanking
                                                        // with averaging for all necessary beams and channels
    unsigned int packetel = mempacket / sizeof(hipfftComplex);
    unsigned char *inbuf = new unsigned char[packetel];
    memset(&hints, 0, sizeof(hints));
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_DGRAM;
    hints.ai_flags = AI_PASSIVE;    // allows to use NULL in getaddrinfo

    cout << "Will process " << memsize / 1024.0 << "KB chunks\n"
		<< "Divided into " << packets << " 6144B packets\n";

    if((rv = getaddrinfo(NULL, PORT, &hints, &servinfo)) != 0) {
        cout << "getaddrinfo error " << gai_strerror(rv) << endl;
        exit(EXIT_FAILURE);
    }

    // loop through the linked list and try binding to the first possible socket
    for (p = servinfo; p != NULL; p = p->ai_next) {
        if((sfd = socket(p->ai_family, p->ai_socktype, p->ai_protocol)) == -1) {
            perror("socket ");
            continue;
        }

        if(bind(sfd, p->ai_addr, p->ai_addrlen) == -1) {
            close(sfd);
            perror("bind ");
            continue;
        }
        break;
    }
    // didn't bind to anything
    if (p == NULL) {
        cout << "error: failed to bind the socket\n";
        exit(EXIT_FAILURE);
    }

    freeaddrinfo(servinfo);     // no longer need it
    cout << "Waiting to receive from the server...\n";

    int chunkno{0};

    header_s head;

    while(chunkno < chunks) {
        // will only receive 6 or 7 channels in one packet
        // will have to stitch them together
        for (unsigned int packetno  = 0; packetno < packets; packetno++) {
            if((numbytes = recvfrom(sfd, inbuf, mempacket, 0, (struct sockaddr*)&their_addr, &addrlen)) == -1 ) {
                cout << "error recvfrom" << endl;
                exit(EXIT_FAILURE);
            }
            // get the vdif header and strip it off the data
            get_header(inbuf, head);
            //cout << "Received packet " << packetno << " with " << numbytes << " bytes\n";
            //cout.flush();
            // I am not happy with the amount of copying done here and below
            // COMMENTED OUT FOR COMPILATION - READING VDIF FILES WILL BE SORTED OUT
            //std::copy(inbuf, inbuf + packetel, chunkbuf + packetno * packetel);
        }

        mypool.add_data(chunkbuf);
        //cout << "Received chunk " << chunkno << endl;
        //cout.flush();
        chunkno++;
        // will send 0 bytes as a last packet to end the loop
        if(!numbytes)
            break;
        inet_ntop(their_addr.ss_family, get_addr((sockaddr*)&their_addr), s, sizeof(s));
    }

    if(test) {

        cout << "Test buffer\n";
        cout.flush();
        // sleep just in case processing is slow
        std::this_thread::sleep_for(std::chrono::seconds(1));

        hipfftComplex *testbuf = new hipfftComplex[batchs * ffts * times * chunks];

        unsigned int seed = std::chrono::system_clock::now().time_since_epoch().count();
        std::mt19937_64 bufeng{seed};
        std::normal_distribution<float> bufdis(0.0, 1.0);

        cout << "Filling the test array...\n";

        for (int ii = 0; ii < batchs * ffts * times * chunks; ii++) {
            testbuf[ii].x = bufdis(bufeng);
            testbuf[ii].y = bufdis(bufeng);
        }

        chunkno = 0;
        while(chunkno < chunks) {
            mypool.add_data(testbuf + chunkno * batchs * ffts * times);
            chunkno++;
        }

    }

    std::this_thread::sleep_for(std::chrono::seconds(2));

    hipDeviceReset();

    return 0;
}
