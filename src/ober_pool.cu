#include <algorithm>
#include <bitset>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <fstream>
#include <memory>
#include <sstream>
#include <thread>
#include <utility>
#include <vector>

#include <boost/array.hpp>
#include <boost/asio.hpp>
#include <boost/bind.hpp>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <numa.h>
#include <pthread.h>
#include <thrust/device_vector.h>

#include "buffer.cuh"
#include "config.hpp"
#include "dedisp/dedisp.hpp"
#include "dedisp/DedispPlan.hpp"
#include "errors.hpp"
#include "filterbank.hpp"
#include "get_mjd.hpp"
#include "gpu_pool.cuh"
#include "heimdall/pipeline.hpp"
#include "kernels.cuh"
#include "ober_pool.cuh"
#include "paf_metadata.hpp"
#include "pdif.hpp"

#include <inttypes.h>
#include <errno.h>
#include <netdb.h>
#include <arpa/inet.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <sys/wait.h>
#include <unistd.h>
#include <signal.h>

using std::thread;
using std::unique_ptr;
using std::vector;

/* ########################################################
TODO: Too many copies - could I use move in certain places?
#########################################################*/

Oberpool::Oberpool(InConfig config) : ngpus(config.ngpus)
{
    for (int ii = 0; ii < ngpus; ii++) {
        gpuvector.push_back(unique_ptr<GpuPool>(new GpuPool(ii, config)));
    }

    for (int ii = 0; ii < ngpus; ii++) {
        threadvector.push_back(thread(&GpuPool::execute, std::move(gpuvector[ii])));
    }
}

Oberpool::~Oberpool(void)
{
    for (int ii = 0; ii < ngpus; ii++) {
        threadvector[ii].join();
    }
}
