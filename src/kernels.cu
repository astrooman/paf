#include "hip/hip_runtime.h"
#include <stdio.h>

#include <kernels.cuh>
__device__ float fftfactor = 1.0/32.0 * 1.0/32.0;

__global__ void rearrange(hipTextureObject_t texObj, hipfftComplex * __restrict__ out)
{
    // this is currently the ugliest solution I can think of
    // xidx is the channel number
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * 128;
    int2 word;

    for (int sample = 0; sample < YSIZE; sample++) {
         word = tex2D<int2>(texObj, xidx, yidx + sample);
         //printf("%i ", sample);
         out[xidx * 128 + 7 * yidx + sample].x = static_cast<float>(static_cast<short>(((word.y & 0xff000000) >> 24) | ((word.y & 0xff0000) >> 8)));
         out[xidx * 128 + 7 * yidx + sample].y = static_cast<float>(static_cast<short>(((word.y & 0xff00) >> 8) | ((word.y & 0xff) << 8)));
         out[336 * 128 + xidx * 128 + 7 * yidx + sample].x = static_cast<float>(static_cast<short>(((word.x & 0xff000000) >> 24) | ((word.x & 0xff0000) >> 8)));
         out[336 * 128 + xidx * 128 + 7 * yidx + sample].y = static_cast<float>(static_cast<short>(((word.x & 0xff00) >> 8) | ((word.x & 0xff) << 8)));
    }
}

__global__ void addtime(float *in, float *out, unsigned int jumpin, unsigned int jumpout, unsigned int factort)
{

    // index will tell which 1MHz channel we are taking care or
    // use 1 thread per 1MHz channel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int ch = 0; ch < 27; ch++) {
	// have to restart to 0, otherwise will add to values from previous execution
        out[idx * 27 + ch] = (float)0.0;
        out[idx * 27 + ch + jumpout] = (float)0.0;
        out[idx * 27 + ch + 2 * jumpout] = (float)0.0;
        out[idx * 27 + ch + 3 * jumpout] = (float)0.0;

        for (int t = 0; t < factort; t++) {
            out[idx * 27 + ch] += in[idx * 128 + ch + t * 32];
            //printf("S1 time sum %f\n", out[idx * 27 + ch]);
            out[idx * 27 + ch + jumpout] += in[idx * 128 + ch + t * 32 + jumpin];
            out[idx * 27 + ch + 2 * jumpout] += in[idx * 128 + ch + t * 32 + 2 * jumpin];
            out[idx * 27 + ch + 3 * jumpout] += in[idx * 128 + ch + t * 32 + 3 * jumpin];
        }
    }
}

__global__ void addchannel(float *in, float *out, unsigned int jumpin, unsigned int jumpout, unsigned int factorc) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    out[idx] = (float)0.0;
    out[idx + jumpout] = (float)0.0;
    out[idx + 2 * jumpout] = (float)0.0;
    out[idx + 3 * jumpout] = (float)0.0;

    for (int ch = 0; ch < factorc; ch++) {
        out[idx] += in[idx * factorc + ch];
        out[idx + jumpout] += in[idx * factorc + ch + jumpin];
        out[idx + 2 * jumpout] += in[idx * factorc + ch + 2 * jumpin];
        out[idx + 3 * jumpout] += in[idx * factorc + ch + 3 * jumpin];
    }

    //printf("S1 freq sum %f\n", out[idx]);
}

__global__ void powerscale(hipfftComplex *in, float *out, unsigned int jump)
{

    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
	// offset introduced, jump to the B polarisation data - can cause some slowing down
	int idx2 = idx1 + jump;
    // these calculations assume polarisation is recorded in x,y base
    // i think the if statement is unnecessary as the number of threads for this
    // kernel 0s fftpoint * timeavg * nchans, which is exactly the size of the output array
    if (idx1 < jump) {      // half of the input data
        float power1 = (in[idx1].x * in[idx1].x + in[idx1].y * in[idx1].y) * fftfactor;
        float power2 = (in[idx2].x * in[idx2].x + in[idx2].y * in[idx2].y) * fftfactor;
        out[idx1] = (power1 + power2); // I; what was this doing here? / 2.0;
        //printf("Input numbers for %i and %i with jump %i: %f %f %f %f, with power %f\n", idx1, idx2, jump, in[idx1].x, in[idx1].y, in[idx2].x, in[idx2].y, out[idx1]);
        out[idx1 + jump] = (power1 - power2); // Q
        out[idx1 + 2 * jump] = 2 * fftfactor * (in[idx1].x * in[idx2].x + in[idx1].y * in[idx2].y); // U
        out[idx1 + 3 * jump] = 2 * fftfactor * (in[idx1].x * in[idx2].y - in[idx1].y * in[idx2].x); // V
    }
}
