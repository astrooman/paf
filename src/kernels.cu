#include "hip/hip_runtime.h"
#include <kernels.cuh>

__global__ void poweradd(hipfftComplex *in, unsigned char *out, unsigned int jump)
{
    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
	// offset introduced - can cause some slowing down
	int idx2 = blockIdx.x * blockDim.x + threadIdx.x + jump;

    if (idx1 < jump) {      // half of the input data
        float power1 = in[idx1].x * in[idx1].x + in[idx1].y * in[idx1].y;
        float power2 = in[idx2].x * in[idx2].x + in[idx2].y * in[idx2].y;
        out[idx1] = (power1 + power2) / 2.0;
    }
}0
