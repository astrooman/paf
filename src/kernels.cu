#include "hip/hip_runtime.h"
#include <stdio.h>

#include <kernels.cuh>

#define XSIZE 7
#define YSIZE 128
#define ZSIZE 48

// __restrict__ tells the compiler there is no memory overlap

__device__ float fftfactor = 1.0/32.0 * 1.0/32.0;

__global__ void rearrange(hipTextureObject_t texObj, hipfftComplex * __restrict__ out)
{
    // this is currently the ugliest solution I can think of
    // xidx is the channel number
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * 128;
    int2 word;
    //if ((xidx == 0) && (yidx == 0)) printf("In the rearrange kernel\n");
    for (int sample = 0; sample < YSIZE; sample++) {
         word = tex2D<int2>(texObj, xidx, yidx + sample);
         printf("%i ", sample);
         out[xidx * 128 + 7 * yidx + sample].x = static_cast<float>(static_cast<short>(((word.y & 0xff000000) >> 24) | ((word.y & 0xff0000) >> 8)));
         out[xidx * 128 + 7 * yidx + sample].y = static_cast<float>(static_cast<short>(((word.y & 0xff00) >> 8) | ((word.y & 0xff) << 8)));
         out[336 * 128 + xidx * 128 + 7 * yidx + sample].x = static_cast<float>(static_cast<short>(((word.x & 0xff000000) >> 24) | ((word.x & 0xff0000) >> 8)));
         out[336 * 128 + xidx * 128 + 7 * yidx + sample].y = static_cast<float>(static_cast<short>(((word.x & 0xff00) >> 8) | ((word.x & 0xff) << 8)));
    }
}

__global__ void rearrange2(hipTextureObject_t texObj, hipfftComplex * __restrict__ out, unsigned int acc)
{

    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
    int yidx = blockIdx.y * 128;
    int chanidx = threadIdx.x + blockIdx.y * 7;
    int skip;
    int2 word;

    for (int ac = 0; ac < acc; ac++) {
        skip = 336 * 128 * 2 * ac;
        for (int sample = 0; sample < YSIZE; sample++) {
            word = tex2D<int2>(texObj, xidx, yidx + ac * 48 * 128 + sample);
            out[skip + chanidx * YSIZE * 2 + sample].x = static_cast<float>(static_cast<short>(((word.y & 0xff000000) >> 24) | ((word.y & 0xff0000) >> 8)));
            out[skip + chanidx * YSIZE * 2 + sample].y = static_cast<float>(static_cast<short>(((word.y & 0xff00) >> 8) | ((word.y & 0xff) << 8)));
            out[skip + chanidx * YSIZE * 2 + YSIZE + sample].x = static_cast<float>(static_cast<short>(((word.x & 0xff000000) >> 24) | ((word.x & 0xff0000) >> 8)));
            out[skip + chanidx * YSIZE * 2 + YSIZE + sample].y = static_cast<float>(static_cast<short>(((word.x & 0xff00) >> 8) | ((word.x & 0xff) << 8)));
        }
    }
}


__global__ void addtime(float *in, float *out, unsigned int jumpin, unsigned int jumpout, unsigned int factort)
{

    // index will tell which 1MHz channel we are taking care or
    // use 1 thread per 1MHz channel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //if (idx == 0) printf("In the time kernel\n");

    for(int ch = 0; ch < 27; ch++) {
	// have to restart to 0, otherwise will add to values from previous execution
        out[idx * 27 + ch] = (float)0.0;
        out[idx * 27 + ch + jumpout] = (float)0.0;
        out[idx * 27 + ch + 2 * jumpout] = (float)0.0;
        out[idx * 27 + ch + 3 * jumpout] = (float)0.0;

        for (int t = 0; t < factort; t++) {
            out[idx * 27 + ch] += in[idx * 128 + ch + t * 32];
            //printf("S1 time sum %f\n", out[idx * 27 + ch]);
            out[idx * 27 + ch + jumpout] += in[idx * 128 + ch + t * 32 + jumpin];
            out[idx * 27 + ch + 2 * jumpout] += in[idx * 128 + ch + t * 32 + 2 * jumpin];
            out[idx * 27 + ch + 3 * jumpout] += in[idx * 128 + ch + t * 32 + 3 * jumpin];
        }
    }
}

/*__global__ void addtime(float* __restrict__ int, float* __restrict__ out, unsigned int jumpin, unsigned int jumpout, unsigned int factort)
{


} */

__global__ void addchannel(float* __restrict__ in, float* __restrict__ out, unsigned int jumpin, unsigned int jumpout, unsigned int factorc) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //if (idx == 0) printf("In the channel kernel\n");

    out[idx] = (float)0.0;
    out[idx + jumpout] = (float)0.0;
    out[idx + 2 * jumpout] = (float)0.0;
    out[idx + 3 * jumpout] = (float)0.0;

    for (int ch = 0; ch < factorc; ch++) {
        out[idx] += in[idx * factorc + ch];
        out[idx + jumpout] += in[idx * factorc + ch + jumpin];
        out[idx + 2 * jumpout] += in[idx * factorc + ch + 2 * jumpin];
        out[idx + 3 * jumpout] += in[idx * factorc + ch + 3 * jumpin];
    }

    //printf("S1 freq sum %f\n", out[idx]);
}

__global__ void addchannel2(float* __restrict__ in, float** __restrict__ out, short nchans, size_t gulp, size_t totsize,  short gulpno, unsigned int jumpin, unsigned int factorc, unsigned int framet, unsigned int acc) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int extra = totsize - gulpno * gulp;
    // thats the starting save position for the chunk of length acc time samples
    int saveidx;

    int inskip;

    for (int ac = 0; ac < acc; ac++) {
        saveidx = (framet % (gulpno * gulp)) * nchans + idx;
        inskip = ac * 27 * 336;
        
        out[0][saveidx] = (float)0.0;
        out[1][saveidx] = (float)0.0;
        out[2][saveidx] = (float)0.0;
        out[3][saveidx] = (float)0.0;

        if ((framet % (gulpno * gulp)) >= extra) {
            for (int ch = 0; ch < factorc; ch++) {
                out[0][saveidx] += in[inskip + idx * factorc + ch];
                out[1][saveidx] += in[inskip + idx * factorc + ch + jumpin];
                out[2][saveidx] += in[inskip + idx * factorc + ch + 2 * jumpin];
                out[3][saveidx] += in[inskip + idx * factorc + ch + 3 * jumpin];
            }
        } else {
            for (int ch = 0; ch < factorc; ch++) {
                out[0][saveidx] += in[inskip + idx * factorc + ch];
                out[1][saveidx] += in[inskip + idx * factorc + ch + jumpin];
                out[2][saveidx] += in[inskip + idx * factorc + ch + 2 * jumpin];
                out[3][saveidx] += in[inskip + idx * factorc + ch + 3 * jumpin];
            }
            // save in two places -save in the extra bit 
            out[0][saveidx + (gulpno * gulp * nchans)] = out[0][saveidx];
            out[1][saveidx + (gulpno * gulp * nchans)] = out[1][saveidx];
            out[2][saveidx + (gulpno * gulp * nchans)] = out[2][saveidx];
            out[3][saveidx + (gulpno * gulp * nchans)] = out[3][saveidx];
            }
        framet++;
    }
    // not a problem - earch thread in a warp uses the same branch
/*    if ((framet % totsize) < gulpno * gulp) {
        for (int ac = 0; ac < acc; ac++) {
            inskip = ac * 27 * 336;
            outskip = ac * 27 * 336 / factorc;
            for (int ch = 0; ch < factorc; ch++) {
                out[0][outskip + saveidx] += in[inskip + idx * factorc + ch];
                out[1][outskip + saveidx] += in[inskip + idx * factorc + ch + jumpin];
                out[2][outskip + saveidx] += in[inskip + idx * factorc + ch + 2 * jumpin];
                out[3][outskip + saveidx] += in[inskip + idx * factorc + ch + 3 * jumpin];
            }
        }
    } else {
        for (int ac = 0; ac < acc; ac++) {
            for (int ch = 0; ch < factorc; ch++) {
                out[0][outskip + saveidx] += in[idx * factorc + ch];
                out[1][outskip + saveidx] += in[idx * factorc + ch + jumpin];
                out[2][outskip + saveidx] += in[idx * factorc + ch + 2 * jumpin];
                out[3][outskip + saveidx] += in[idx * factorc + ch + 3 * jumpin];
            }
            // save in two places - wrap wround to the start of the buffer
            out[0][outskip + saveidx - (gulpno * gulp * nchans)] = out[0][outskip + saveidx];
            out[1][outskip + saveidx - (gulpno * gulp * nchans)] = out[1][outskip + saveidx];
            out[2][outskip + saveidx - (gulpno * gulp * nchans)] = out[2][outskip + saveidx];
            out[3][outskop + saveidx - (gulpno * gulp * nchans)] = out[3][outskip + saveidx];
        }
    }
*/
}

__global__ void powerscale(hipfftComplex *in, float *out, unsigned int jump)
{

    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    //if (idx1 == 0) printf("In the power kernel\n");
    // offset introduced, jump to the B polarisation data - can cause some slowing down
    int idx2 = idx1 + jump;
    // these calculations assume polarisation is recorded in x,y base
    // i think the if statement is unnecessary as the number of threads for this
    // kernel 0s fftpoint * timeavg * nchans, which is exactly the size of the output array
    if (idx1 < jump) {      // half of the input data
        float power1 = (in[idx1].x * in[idx1].x + in[idx1].y * in[idx1].y) * fftfactor;
        float power2 = (in[idx2].x * in[idx2].x + in[idx2].y * in[idx2].y) * fftfactor;
        out[idx1] = (power1 + power2); // I; what was this doing here? / 2.0;
        //printf("Input numbers for %i and %i with jump %i: %f %f %f %f, with power %f\n", idx1, idx2, jump, in[idx1].x, in[idx1].y, in[idx2].x, in[idx2].y, out[idx1]);
        out[idx1 + jump] = (power1 - power2); // Q
        out[idx1 + 2 * jump] = 2 * fftfactor * (in[idx1].x * in[idx2].x + in[idx1].y * in[idx2].y); // U
        out[idx1 + 3 * jump] = 2 * fftfactor * (in[idx1].x * in[idx2].y - in[idx1].y * in[idx2].x); // V
    }
}

__global__ void powertime(hipfftComplex* __restrict__ in, float* __restrict__ out, unsigned int jump, unsigned int factort)
{
    // 1MHz channel ID
    int idx1 = blockIdx.x;
    // 'small' channel ID
    int idx2 = threadIdx.x;
    float power1;
    float power2;

    idx1 = idx1 * YSIZE * 2;
    int outidx = 27 * blockIdx.x + threadIdx.x;

    out[outidx] = (float)0.0;
    out[outidx + jump] = (float)0.0;
    out[outidx + 2 * jump] = (float)0.0;
    out[outidx + 3 * jump] = (float)0.0;

    for (int ii = 0; ii < factort; ii++) {
        idx2 = threadIdx.x + ii * 32;
	power1 = (in[idx1 + idx2].x * in[idx1 + idx2].x + in[idx1 + idx2].y * in[idx1 + idx2].y) * fftfactor;
        power2 = (in[idx1 + 128 + idx2].x * in[idx1 + 128 + idx2].x + in[idx1 + 128 + idx2].y * in[idx1 + 128 + idx2].y) * fftfactor;
	out[outidx] += (power1 + power2);
        out[outidx + jump] += (power1 - power2);
        out[outidx + 2 * jump] += (2 * fftfactor * (in[idx1 + idx2].x * in[idx1 + 128 + idx2].x + in[idx1 + idx2].y * in[idx1 + 128 + idx2].y));
        out[outidx + 3 * jump] += (2 * fftfactor * (in[idx1 + idx2].x * in[idx1 + 128 + idx2].y - in[idx1 + idx2].y * in[idx1 + 128 + idx2].x));

    }

   printf("%i, %i: %i\n", blockIdx.x, threadIdx.x, out[outidx]);
}

__global__ void powertime2(hipfftComplex* __restrict__ in, float* __restrict__ out, unsigned int jump, unsigned int factort, unsigned int acc) {

    int idx1, idx2;
    int outidx;
    int skip1, skip2;
    float power1, power2;

    for (int ac = 0; ac < acc; ac++) {
        skip1 = ac * 336 * 128 * 2;
        skip2 = ac * 336 * 27;
        for (int ii = 0; ii < 7; ii++) {
            outidx = skip2 + 7 * 27 * blockIdx.x + ii * 27 + threadIdx.x;
            out[outidx] = (float)0.0;
            out[outidx + jump] = (float)0.0;
            out[outidx + 2 * jump] = (float)0.0;
            out[outidx + 3 * jump] = (float)0.0;

            idx1 = skip1 + 256 * (blockIdx.x * 7 + ii);

            for (int jj = 0; jj < factort; jj++) {
                idx2 = threadIdx.x + jj * 32;
                power1 = (in[idx1 + idx2].x * in[idx1 + idx2].x + in[idx1 + idx2].y * in[idx1 + idx2].y) * fftfactor;
                power2 = (in[idx1 + 128 + idx2].x * in[idx1 + 128 + idx2].x + in[idx1 + 128 + idx2].y * in[idx1 + 128 + idx2].y) * fftfactor;
        	out[outidx] += (power1 + power2);
                out[outidx + jump] += (power1 - power2);
                out[outidx + 2 * jump] += (2 * fftfactor * (in[idx1 + idx2].x * in[idx1 + 128 + idx2].x + in[idx1 + idx2].y * in[idx1 + 128 + idx2].y));
                out[outidx + 3 * jump] += (2 * fftfactor * (in[idx1 + idx2].x * in[idx1 + 128 + idx2].y - in[idx1 + idx2].y * in[idx1 + 128 + idx2].x));
            }
        }
    }

//    printf("%i, %i: %i\n", blockIdx.x, threadIdx.x, out[outidx]);
}

__global__ void scale(float* in, float* out, unsigned int nchans, unsigned int time_samples)
{
    // call one block with 32 threads
    // be careful when processing total sizes that cannot be divided by 32
    // or make sure the total size can be divided by 32 when allocating
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float nrec = 1.0f / (float)nchans;
    float mean;
    float std;

    unsigned int threads = blockDim.x * gridDim.x;
    unsigned int start = 0;
    float nrec1 = 1.0f / (float)(nchans -1.0f);
    for (int chunk = 0; chunk < (time_samples / threads); chunk++) {
        mean = 0.0f;
        std = 0.0f;
        start = chunk * threads * nchans;

        for (int ii = 0; ii < nchans; ii++) {
            mean += in[start + idx * nchans + ii] * nrec;
            //printf("%d\n", mean);
        }

        for (int jj = 0; jj < nchans; jj++) {
            std += (in[start + idx * nchans + jj] - mean) * (in[start + idx * nchans + jj] - mean);
        }
        std *= nrec1;

        //printf("%i: %i, %f, %f, %f\n", idx, nchans, nrec, mean, std);

        float stdrec = rsqrtf(std);

        for (int kk = 0; kk < nchans; kk++) {
            out[start + idx * nchans + kk] = ((in[start + idx * nchans + kk] - mean) * stdrec) * 32.0f + 64.0f;
            if (out[start + idx * nchans + kk] < 0.0f)
                out[start + idx * nchans + kk] = 0.0f;
        }
    }
}
