#include <cstdlib>
#include <exception>
#include <iostream>
#include <sys/stat.h>
#include <sstream>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "config.hpp"
#include "errors.hpp"
#include "main_pool.cuh"

using std::cerr;
using std::cout;
using std::endl;
using std::exception;
using std::string;
using std::thread;
using std::vector;

int main(int argc, char *argv[])
{
    string configfile;
    InConfig config;
    SetDefaultConfig(config);

    // too many parameters to load as arguments - use config file
    if (argc >= 2) {
        for (int iarg = 0; iarg < argc; iarg++) {
            if (string(argv[iarg]) == "--config") {      // configuration file
                iarg++;
                configfile = string(argv[iarg]);
                try {
                    ReadConfig(configfile, config);
                } catch (const exception &exc) {
                    cout << exc.what() << endl;
                    // NOTE: Currently can to nothing with certain values missing from the config file.
                    // TODO: Make sure that the pipeline can run with defaul configuration only.
                    return 1;
                }
            } else if (string(argv[iarg]) == "-r") {
                iarg++;
                config.record = atoi(argv[iarg]);
            } else if (string(argv[iarg]) == "-s") {
                iarg++;
                config.scaleseconds = atof(argv[iarg]);
            } else if (string(argv[iarg]) == "-b") {     // the number of beams to accept the data from
                iarg++;
                config.nobeams = atoi(argv[iarg]);
            } else if (string(argv[iarg]) == "-t") {     // the number of time sample to average
                iarg++;
                config.timeavg = atoi(argv[iarg]);
            } else if (string(argv[iarg]) == "-f") {     // the number of frequency channels to average
                iarg++;
                config.freqavg = atoi(argv[iarg]);
            } else if (string(argv[iarg]) == "-n") {    // the number of GPUs to use
                iarg++;
                config.nogpus = atoi(argv[iarg]);
                int devcount{0};
                cudaCheckError(hipGetDeviceCount(&devcount));
                if (config.nogpus > devcount) {
                    cout << "You can't use more GPUs than you have available!" << endl;
                    config.nogpus = devcount;
                }
            } else if (string(argv[iarg]) == "-o") {    // output directory for the filterbank files
                iarg++;
                struct stat chkdir;
                if (stat(argv[iarg], &chkdir) == -1) {
                    cerr << "Stat error" << endl;
                } else {
                    bool isdir = S_ISDIR(chkdir.st_mode);
                    if (isdir)
                        config.outdir = string(argv[iarg]);
                    else
                        cout << "Output directory does not exist! Will use the default directory!";
                }
            } else if (string(argv[iarg]) == "-k") {
                iarg++;
                std::stringstream sskey;
                sskey << std::hex << argv[iarg];
                sskey >> config.dadakey;
            } else if (string(argv[iarg]) == "--gpuid") {
                for (int igpu = 0; igpu < config.nogpus; igpu++) {
                    iarg++;
                    config.gpuids.push_back(atoi(argv[iarg]));
                }
            } else if (string(argv[iarg]) == "--ip") {
                for (int iip = 0; iip < config.nogpus; iip++) {
                    iarg++;
                    config.ips.push_back(string(argv[iarg]));
                }
            } else if (string(argv[iarg]) == "--numa") {
                iarg++;
                config.numa = atoi(argv[iarg]);
            } else if (string(argv[iarg]) == "-v") {
                config.verbose = true;
            } else if ((string(argv[iarg]) == "-h") || (string(argv[iarg]) == "--help")) {
                cout << "Options:\n"
                        << "\t -h --help - print out this message\n"
                        << "\t --config <file name> - configuration file\n"
                        << "\t -b - the number of beams to process\n"
                        << "\t -f - the number of frequency channels to average\n"
                        << "\t -n - the number of GPUs to use\n"
                        << "\t -o <directory> - output directory\n"
                        << "\t -k DADA key to use\n"
                        << "\t -r - the number of seconds to record\n"
                        << "\t -s - the number of seconds to record for scaling factors\n"
                        << "\t -t - the number of time samples to average\n"
                        << "\t -v - use verbose mode\n"
                        << "\t --gpuid - GPU IDs to use - the number must be the same as 'n'\n"
                        << "\t --ip - IPs to listen to - the number must be the same as 'n'\n\n";
                exit(EXIT_SUCCESS);
            }
        }

    }

    if (config.verbose) {
        cout << "Starting up. This may take few seconds..." << endl;
        PrintConfig(config);
    }

    MainPool pafpool(config);

    return 0;
}
