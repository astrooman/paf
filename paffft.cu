#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

using std::cerr;
using std::cout;
using std::endl;

int main(int argc, char* argv[])
{



    const unsigned int arrsize = 32;
    const unsigned int fftsize = arrsize;
    const unsigned int batchsize = 1152;    // the number of FFTs we want to perform at once
    hipfftComplex *h_inarray = new hipfftComplex[arrsize];

    cout << "Pre-initialisation...\n";

    // this should make the first proper FFT execution faster
    cufftHanfle preinit;
    hipfftPlan1d(&preinit, fftsize, HIPFFT_C2C, 1);

    unsigned long seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::mt19937_64 arreng{seed};
    std::normal_distribution<float> arrdis(0.0, 1.0);

    for (int ii = 0; ii < arrsize; ii++) {
            h_inarray[ii].x = arrdis(arreng);
            h_inarray[ii].y = arrdis(arreng);
    }

    hipfftComplex *d_inarray = new hipfftComplex[arrsize];
    hipMalloc((void**)&d_inarray, sizeof(hipfftComplex) * arrsize);
    hipMemcpy(d_inarray, h_inarray, sizeof(hipfftComplex) * arrsize, hipMemcpyHostToDevice);

    cout << "Performing single FFT...\n";

    hipfftHandle singleplan
    hipfftPlan1d(&singleplan, fftsize, HIPFFT_C2C, 1);
    hipfftExecC2C(singleplan, d_inarray, d_inarray, HIPFFT_FORWARD);


    return 0;

}
