#include <chrono>
#include <cmath>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <nvToolsExt.h>

using std::cerr;
using std::cout;
using std::endl;


static const uint32_t colors[] = { 0x0000ff00, 0x000000ff, 0x00ffff00, 0x00ff00ff, 0x0000ffff, 0x00ff0000, 0x00ffffff };
static const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
	int color_id = cid; \
	color_id = color_id%num_colors;\
	nvtxEventAttributes_t eventAttrib = {0}; \
	eventAttrib.version = NVTX_VERSION; \
	eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
	eventAttrib.colorType = NVTX_COLOR_ARGB; \
	eventAttrib.color = colors[color_id]; \
	eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
	eventAttrib.message.ascii = name; \
	nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();

int main(int argc, char* argv[])
{



    const unsigned int arrsize = 32;
    const unsigned int fftsize = arrsize;
    const unsigned int batchsize = 1152;    // the number of FFTs we want to perform at once
    hipfftComplex *h_inarray = new hipfftComplex[arrsize];


    cout << "Pre-initialisation...\n";

    PUSH_RANGE("FFT pre-init", 0)
    // this should make the first proper FFT execution faster
    hipfftHandle preinit;
    hipfftPlan1d(&preinit, fftsize, HIPFFT_C2C, 1);
    POP_RANGE

    unsigned long seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::mt19937_64 arreng{seed};
    std::normal_distribution<float> arrdis(0.0, 1.0);

    for (int ii = 0; ii < arrsize; ii++) {
            h_inarray[ii].x = arrdis(arreng);
            h_inarray[ii].y = arrdis(arreng);
    }

    hipfftComplex *d_inarray = new hipfftComplex[arrsize];
    hipMalloc((void**)&d_inarray, sizeof(hipfftComplex) * arrsize);
    hipMemcpy(d_inarray, h_inarray, sizeof(hipfftComplex) * arrsize, hipMemcpyHostToDevice);

    cout << "Performing single FFT...\n";

    PUSH_RANGE("Single FFT init", 1)
    hipfftHandle singleplan;
    hipfftPlan1d(&singleplan, fftsize, HIPFFT_C2C, 1);
    POP_RANGE

    PUSH_RANGE("Single FFT exec", 2)
    hipfftExecC2C(singleplan, d_inarray, d_inarray, HIPFFT_FORWARD);
    POP_RANGE

    hipfftDestroy(singleplan);
    hipFree(d_inarray);
    delete [] h_inarray;

    hipfftComplex *h_inarraym = new hipfftComplex[arrsize * batchsize];

    for (int ii = 0; ii < arrsize * batchsize; ii++) {
        h_inarraym[ii].x = arrdis(arreng);
        h_inarraym[ii].y = arrdis(arreng);

    }

    hipfftComplex *d_inarraym = new hipfftComplex[arrsize * batchsize];
    hipMalloc((void**)&d_inarraym, sizeof(hipfftComplex) * arrsize * batchsize);
    hipMemcpy(d_inarraym, h_inarraym, sizeof(hipfftComplex) * arrsize * batchsize, hipMemcpyHostToDevice);

    cout << "Performing multi FFT...\n";

    int sizes[1] = {fftsize};

    PUSH_RANGE("Multi FFT init", 3)
    hipfftHandle multiplan;
    hipfftPlanMany(&multiplan, 1, sizes, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batchsize);
    POP_RANGE

    PUSH_RANGE("Multi FFT exec", 4)
    hipfftExecC2C(multiplan, d_inarraym, d_inarraym, HIPFFT_FORWARD);
    POP_RANGE

    hipfftDestroy(multiplan);
    hipFree(d_inarraym);
    delete [] h_inarraym;

    return 0;

}
